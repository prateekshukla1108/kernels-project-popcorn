
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define N 10      // a preprocessor macro for defining the number of elements in array

// simple CUDA kernel for ReLU activation function
__global__ void ReLU(float *vec, float *res, int size){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        res[idx] =  vec[idx] > 0 ? vec[idx] : 0;
    }
}

void printArray(float *arr, int size){
    for(int i=0; i<size; i++){
        printf("%f ", arr[i]);
    }printf("\n");
}
int main(){
    size_t bytes = N * sizeof(float);    

    float *vec_h, *res_h;
    
    // allocating the the memory
    vec_h = (float*)malloc(bytes);
    res_h = (float*)malloc(bytes);

    // initializing the vector
    srand(time(NULL));
    for(int i=0; i<N; i++){
        vec_h[i] = ((float)rand() / (float)(RAND_MAX)) * 50.0f - 25.0f;  // Generate a random float between -25.0 and 25.0
    }

    float *vec_d, *res_d;
    hipMalloc((void**)&vec_d,bytes);
    hipMalloc((void**)&res_d, bytes);

    hipMemcpy(vec_d, vec_h, bytes, hipMemcpyHostToDevice);

    // defining the blockDim and gridDim
    dim3 THREADS(32, 1, 1);
    dim3 BLOCKS(N + THREADS.x / float(THREADS.x), 1, 1);

    // launching the kernel
    ReLU<<<BLOCKS, THREADS>>>(vec_d, res_d, N);

    hipMemcpy(res_h, res_d, bytes, hipMemcpyDeviceToHost);

    printf("vector:\n");
    printArray(vec_h, N);
    printf("vector after ReLU:\n");
    printArray(res_h, N);
    return 0;
}
