
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(float* A, float* B, float* C, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000;
    int size = N * sizeof(float);
    
    // host arrays
    float *A = new float[N], *B = new float[N], *C = new float[N];

    // define A and B
    for (int i = 0; i < N; ++i) {
        A[i] = i * 1.0f;       
        B[i] = i * 2.0f;       // double of A
    }

    // device arrays
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // launch kernel, ceiling division for number of blocks
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C, N);

    // copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // print first 10 results
    for (int i = 0; i < 10; ++i) {
        std::cout << "C[" << i << "] = " << C[i] << std::endl;
    }

    // free memory
    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
