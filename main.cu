#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kernels/vectorized-layernorm.cu"

int main(){

    int M = 1024;
    int N = 1024;

    size_t matrix_size = M*N*sizeof(float);
    float *X_input, *P_output;
    float *D_input, *D_output;

    X_input = (float*)malloc(matrix_size);
    P_output = (float*)malloc(matrix_size);

    for(int i = 0; i < M*N; i++){
        X_input[i] = i+1;
    }

    hipMalloc((void**)&D_input, matrix_size);
    hipMalloc((void**)&D_output, matrix_size);

    hipMemcpy(D_input, X_input, matrix_size, hipMemcpyHostToDevice);

    run_vect_ln(D_input, D_output, M, N);

    hipMemcpy(P_output, D_output, matrix_size, hipMemcpyDeviceToHost);

    printf("Input matrix: \n");
    for(int i=0; i<5; i++){
        for(int j=0; j<5; j++){
            printf("%f", X_input[i*N+j]);
        }
        printf("\n");
    }

    printf("Output matrix: \n");
    for(int i=0; i<5; i++){
        for(int j=0; j<5; j++){
            printf("%f", P_output[i*N+j]);
        }
        printf("\n");
    }

    free(P_output); free(X_input);
    hipFree(D_input); hipFree(D_output);

}