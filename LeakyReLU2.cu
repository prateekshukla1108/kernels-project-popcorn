#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 1024
#define THREADS_PER_BLOCK 256

__global__ void Leaky_ReLU(float *input, float *output, int size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N){
        output[idx] = fmaxf(0.01f * input[idx], input[idx]);
    }
}

int main()
{
    float *d_input, *h_input, *d_output, *h_output;
    size_t size = N * sizeof(float);

    h_input = (float *)malloc(size);
    h_output = (float *)malloc(size);

    for (int i = 0; i < N; i++){
        h_input[i] = (float)i - N / 2;
    }

    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    int block_size = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    
    Leaky_ReLU<<<block_size, THREADS_PER_BLOCK>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    // Verify results
    for (int i = 0; i < N; i++) {
        printf("LeakyReLU(%f) = %f\n", h_input[i], h_output[i]);
    }

    // Clean up
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
    

}
