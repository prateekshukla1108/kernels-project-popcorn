
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a = 5, b = 3, c;
    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    hipMalloc(&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add<<<1, 256>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Result: " << c << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}