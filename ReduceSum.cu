#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

#define BLOCK_SIZE 256

__global__ void reduceSumKernel(float *d_input, float *d_output, int N) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory
    sdata[tid] = (i < N) ? d_input[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Warp-level reduction (optional)
    // if (tid < 32) {
    //     for (int s = 16; s > 0; s >>= 1) {
    //         sdata[tid] += sdata[tid + s];
    //     }
    // }

    // Write result to output
    if (tid == 0) {
        d_output[blockIdx.x] = sdata[0];
    }
}

int main() {
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    // host memory
    float *h_input;
    float *h_output;


    hipHostMalloc(&h_input, size, hipHostMallocDefault);
    hipHostMalloc(&h_output, sizeof(float), hipHostMallocDefault);

    // Init with random
    srand(time(0));
    for (int i = 0; i < N; i++) {
        h_input[i] = static_cast<float>(rand() % 10 + 1); 
    }

    // device memory
    float *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, sizeof(float));

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // temporary storage for block sums
    float *d_blockSums;
    hipMalloc(&d_blockSums, gridSize * sizeof(float));

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Reduce within blocks
    reduceSumKernel<<<gridSize, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(d_input, d_blockSums, N);

    // Reduce block sums
    reduceSumKernel<<<1, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(d_blockSums, d_output, gridSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "GPU reduction time: " << elapsedTime << " ms" << endl;

    hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    cout << "Sum: " << h_output[0] << endl;

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_blockSums);

    hipHostFree(h_input);
    hipHostFree(h_output);

    // dont forget to destroy the events too
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
