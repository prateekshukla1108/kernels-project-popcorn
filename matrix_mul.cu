#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../helpers/cuda_helpers.h"

// Kernel to calculate P = M * N where M and N are square matrices
__global__ void matrixMulKernel(float* M, float* N, float* P, int width){
    int Row = blockDim.x * blockIdx.x + threadIdx.x;
    int Col = blockDim.y * blockIdx.y + threadIdx.y;

    if ((Row < width) && (Col < width)){
        float Pvalue = 0;
        for (int i = 0; i < width; i++){
            // Since here we access element of M and one element of N
            // and we apply one addition and one multiplication.
            // compute to global-memory-access ratio is 1. 
            // But we need to increase this ratio!
            Pvalue += M[Row * width + i] * N[i * width + Col];
        }
        P[Row * width + Col] = Pvalue;
    }
}
