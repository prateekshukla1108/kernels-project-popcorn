#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void add(int *a, int *b, int *c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int N = 16;
    int size = N * sizeof(int);

    // Allocate memory on the host
    int *h_a = (int *)malloc(size);
    int *h_b = (int *)malloc(size);
    int *h_c = (int *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * i;
    }

    // Allocate memory on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define the execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // Copy the result back to the host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        cout << h_c[i] << " ";
    }
    cout << endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}