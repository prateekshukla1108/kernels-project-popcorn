// nvcc vec_cublas.cu -o vec_cublas -lstdc++ -lcublas


#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    // Initialize input vectors (you might want to add your own initialization)
    for(int i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i;
    }

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate device memory
    float *d_a, *d_b;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Scaling factors
    const float alpha = 1.0f;

    // Perform vector addition: C = alpha*A + B
    hipblasSaxpy(handle, N, &alpha, d_a, 1, d_b, 1);

    // Copy result back to host (result is in d_b)
    hipMemcpy(C, d_b, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    for(int i = 0; i < N; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipblasDestroy(handle);

    return 0;
}
