#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void helloKernel() {
    printf("Hello, World from thread %d!\n", threadIdx.x);
}

int main() {
    helloKernel<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}