#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100

__global__ void MatMul(int *A ,int *B, int *C, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < n && j < n){
        int sum = 0;
        for (int k = 0; k < n; k++){
            sum += A[i * n + k] * B[k * n + j];
        }
        C[i * n + j] = sum;
    }
}

int main()
{
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = (int *)malloc(N * N * sizeof(int));
    h_B = (int *)malloc(N * N * sizeof(int));
    h_C = (int *)malloc(N * N * sizeof(int));

    for (int i = 0; i < N; i++){
        for(int j = 0; j < N; j++){
            h_A[i * N + j] = i + j;
            h_B[i * N + j] = i * j;
        }
    }

    hipMalloc((void **)&d_A, N * N * sizeof(int));
    hipMalloc((void **)&d_B, N * N * sizeof(int));
    hipMalloc((void **)&d_C, N * N * sizeof(int));

    hipMemcpy(d_A, h_A, N * N *sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N *sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(4, 4);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    MatMul<<<blockDim, gridDim>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, N * N * sizeof(int), hipMemcpyHostToDevice);
        // Print the result matrix C
    printf("Matrix Multiplication (C = A * B):\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d ", h_C[i * N + j]);
        }
        printf("\n");
    }

    // Free allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
