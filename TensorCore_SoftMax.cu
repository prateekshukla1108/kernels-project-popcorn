#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>  // FP16 support for Tensor Cores

#define BLOCK_SIZE 256
#define WARP_SIZE 32

// Warp reduction for max operation (FP16)
__device__ __half warpReduceMax(__half val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        val = __hmax(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

// Warp reduction for sum operation (FP16)
__device__ __half warpReduceSum(__half val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        val = __hadd(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

// Optimized Softmax Kernel using Tensor Cores (FP16)
__global__ void softmaxTensorCore(__half* input, __half* output, int rows, int cols) {
    extern __shared__ __half shared_data[];

    int row = blockIdx.x;
    int tid = threadIdx.x;

    if (row >= rows) return;

    __half* row_data = input + row * cols;
    __half* shared_row = shared_data;

    // Step 1: Compute max for numerical stability
    __half max_val = __float2half(-INFINITY);
    for (int i = tid; i < cols; i += blockDim.x) {
        max_val = __hmax(max_val, row_data[i]);
    }
    max_val = warpReduceMax(max_val);

    // Step 2: Compute exponentials and sum
    __half sum = __float2half(0.0f);
    for (int i = tid; i < cols; i += blockDim.x) {
        shared_row[i] = __float2half(expf(__half2float(__hsub(row_data[i], max_val))));  // ✅ Corrected
        sum = __hadd(sum, shared_row[i]);
    }
    sum = warpReduceSum(sum);

    // Step 3: Normalize values
    for (int i = tid; i < cols; i += blockDim.x) {
        output[row * cols + i] = __hdiv(shared_row[i], sum);
    }
}

// Host function to launch kernel
void softmaxFP16(float* h_input, float* h_output, int rows, int cols) {
    __half *d_input, *d_output;
    size_t size = rows * cols * sizeof(__half);

    // Allocate device memory
    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    // Convert FP32 input to FP16
    __half* h_input_half = new __half[rows * cols];
    for (int i = 0; i < rows * cols; i++)
        h_input_half[i] = __float2half(h_input[i]);

    // Copy data to device
    hipMemcpy(d_input, h_input_half, size, hipMemcpyHostToDevice);

      // Create events for timing
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
  
      // Record the start time
      hipEventRecord(start);

    // Launch optimized softmax kernel
    int shared_mem_size = cols * sizeof(__half);
    softmaxTensorCore<<<rows, BLOCK_SIZE, shared_mem_size>>>(d_input, d_output, rows, cols);

     // Record the stop time
     hipEventRecord(stop);
     hipEventSynchronize(stop);

      // Calculate execution time in milliseconds
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    float timeInSeconds = elapsedTime / 1000.0f;


    // Copy result back to host
    __half* h_output_half = new __half[rows * cols];
    hipMemcpy(h_output_half, d_output, size, hipMemcpyDeviceToHost);

    // Convert FP16 output to FP32
    for (int i = 0; i < rows * cols; i++)
        h_output[i] = __half2float(h_output_half[i]);

    // Cleanup
    delete[] h_input_half;
    delete[] h_output_half;
    hipFree(d_input);
    hipFree(d_output);
    
    // Calculate GFLOPS
    long long numFLOPs = (long long)rows * cols * 2;  // 2 FLOPs per element: exp and division
    float gflops = (numFLOPs / timeInSeconds) / 1e9;

    // Output the results
    std::cout << "Execution time: " << elapsedTime << " ms\n";
    std::cout << "GFLOPS: " << gflops << " GFLOPS\n";
}

// Test function
int main() {

    const int rows = 2, cols = 4;
    float h_input[rows * cols] = {1.0, 2.0, 3.0, 4.0,  5.0, 6.0, 7.0, 8.0};
    float h_output[rows * cols];

    // Compute softmax using Tensor Core acceleration
    softmaxFP16(h_input, h_output, rows, cols);

    // Print result
    std::cout << "Softmax Output (FP16 Accelerated):\n";
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            std::cout << h_output[i * cols + j] << " ";
        }
        std::cout << "\n";
    }

    return 0;

}
