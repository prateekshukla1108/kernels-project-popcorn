#include <stdio.h>
#include <hip/hip_runtime.h>

#define TW 2


__global__
void mykernel(int *a, int *b, int *c, int N){

    // loading from shared memory
    __shared__ float as[TW][TW];
    __shared__ float bs[TW][TW];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * N + col;

    // Update tiled vectors
    if (row < N and col < N){
        as[threadIdx.x][threadIdx.y] = a[row * N + col];
        bs[threadIdx.x][threadIdx.y] = b[row * N + col];
    }
    __syncthreads();

    // perform addition
    if (row < N and col < N){
        c[row * N + col] = as[threadIdx.x][threadIdx.y] + bs[threadIdx.x][threadIdx.y];
    }
    __syncthreads();

}


int main(){
    int N = 4;
    int *h_a, *h_b, *h_c; // host array
    int *d_a, *d_b, *d_c; // device array

    // acclocate hosty memory
    h_a = (int *)malloc(N * N * sizeof(int));
    h_b = (int *)malloc(N * N * sizeof(int));
    h_c = (int *)malloc(N * N * sizeof(int));

    // Initialize host array
    for (int i = 0; i < N; i ++){
        for (int j = 0; j < N; j++){
            h_a[i * N + j] = i * N + j;
            h_b[i * N + j] = i * N + j;
        }
    }

    // Allocate device memory
    hipMalloc(&d_a, N * N * sizeof(int));
    hipMalloc(&d_b, N * N * sizeof(int));
    hipMalloc(&d_c, N * N * sizeof(int));

    // Copy data to device
    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, N * N * sizeof(int), hipMemcpyHostToDevice);


    // Launch kernel
    dim3 tpd(TW, TW);
    dim3 gridDim((N + TW - 1) / TW, (N + TW -1) / TW);
    // shared size
    mykernel<<<gridDim, tpd>>>(d_a, d_b, d_c, N);

    // copy data from device to host
    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // verify the resultz
    for (int i = 0; i < N; i ++){
        for (int j = 0; j < N; j++){
            printf("h_c[%d][%d] = %d\n", i, j, h_c[i * N + j]);

        }
    }

    // release
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
}
