#include "hip/hip_runtime.h"
#include "helpers.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__
void vectorMultiply(float *d_Output, const float *d_A, const float *d_B, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size){
        d_Output[i] = d_A[i] * d_B[i];
    }
}

__host__
int main(){
    int size = 1000000;
    size_t sizeBytes = size * sizeof(float);

    float *h_A, *h_B, *h_Output;
    float *d_A, *d_B, *d_Output;

    h_A = (float *)malloc(sizeBytes);
    h_B = (float *)malloc(sizeBytes);
    h_Output = (float *)malloc(sizeBytes);

    for(int i = 0; i < size; ++i){
        h_A[i] = i;
        h_B[i] = size - i;
    }

    CUDA_CHECK(hipMalloc((void **)&d_A, sizeBytes));
    CUDA_CHECK(hipMalloc((void **)&d_B, sizeBytes));
    CUDA_CHECK(hipMalloc((void **)&d_Output, sizeBytes));

    CUDA_CHECK(hipMemcpy(d_A, h_A, sizeBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, sizeBytes, hipMemcpyHostToDevice));

    int threadsPerBlock = 512;
    int blocksPerGrid = (size + threadsPerBlock -  1)/threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    vectorMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_Output, d_A, d_B, size);

    hipError_t kernelLaunchCheck = hipGetLastError();
    if (kernelLaunchCheck != hipSuccess) {
        fprintf(stderr, "CUDA Error at %s: %d\n", __FILE__, __LINE__);                  
        fprintf(stderr, "Error code: %d, %s\n", kernelLaunchCheck, hipGetErrorString(kernelLaunchCheck));       
        exit(EXIT_FAILURE);                              
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_Output, d_Output, sizeBytes, hipMemcpyDeviceToHost);

    printf("Verifying Results: \n");
    for (int i = 0; i < 10; ++i){
        printf("a[%d] = %f, b[%d] = %f, a[%d] * b[%d] = %f\n",i, h_A[i], i, h_B[i], i, i, h_Output[i]);
    }

    printf("Time Elapsed (GPU): %.3f seconds \n", milliseconds);

    clock_t start_cpu, stop_cpu;
    double cpu_time_used;
    start_cpu = clock();

    for (int i =0; i < size; i++){
        h_Output[i] = h_A[i] * h_B[i];
    }

    stop_cpu = clock();
    cpu_time_used = ((double)(stop_cpu - start_cpu)) / CLOCKS_PER_SEC * 1000;
    printf("Time Elapsed (CPU): %.3f\n", cpu_time_used);


    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_Output));

    free(h_A);
    free(h_B);
    free(h_Output);

    return 0;
}