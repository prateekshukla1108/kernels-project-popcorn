#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <torch/types.h>

#define CUDA_CHECK(ans)                        \
    {                                          \
        cudaAssert((ans), __FILE__, __LINE__); \
    }
inline void cudaAssert(hipError_t code, const char* file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA error %s: %s at %s: %d\n",
                hipGetErrorName(code), hipGetErrorString(code),
                file, line);
        exit(code);
    }
}
#define CEIL_DIV(x, y) ((x) >= 0 ? (((x) + (y) - 1) / (y)) : ((x) / (y)))
#define PI 3.1415

float random_normal_clamped(float min, float max) {
    float u1 = (float)rand() / RAND_MAX;
    float u2 = (float)rand() / RAND_MAX;
    float num = sqrtf(-2.0f * logf(u1)) * cosf(2.0f * PI * u2);
    if (num < min)
        return min;
    if (num > max)
        return max;
    return num;
}

/*
Reduction functions on device. These will be inline:
The compiler will replace the call with the code instead of calling the function (overhead)
*/
/*
Utility warp level sum reduction with shuffle instructions
*/

__device__ __forceinline__ float warpReduceSum(float val, int width) {
    for (int offset = width / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }

    return val;
}

__device__ __forceinline__ float warpReduceMax(float val, int width) {
    for (int offset = width / 2; offset > 0; offset /= 2) {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset));
    }

    return val;
}

/*
This kernel uses flash attention algorithm to compute multi-head attention.
Q, K, and V are 4D tensors of shape (batch_size, n_heads, seq_len, embed_dim).
Additional inputs are Tr and Tc which are the tiles each block computes.
The arrays l and m are to save the norm and maximum for the ith tile.
SRAM will have size M and Br = ceil(M / 4d) and Bc = min(ceil(M / 4d), d)
where M is the size of the SRAM.
*/
template <const int Br, const int Bc>
__global__ void flash_attn_1_kernel(float* Q, float* K, float* V, int N, int d, int Tr, int Tc, float scale, float* l, float* m, float* O) {
    int tx = threadIdx.x;  // Br * Bc threads

    int bx = blockIdx.x;  // Batch index
    int by = blockIdx.y;  // Head index

    // tip to calculate offset:
    // count how many elements to skip in the array to reach an index
    int qkv_off = (bx * gridDim.y * N * d) + (by * N * d);
    int lm_off = (bx * gridDim.y * N) + (by * N);

    // TODO: remove too much shared memory usage
    extern __shared__ float smem[];
    float* Qi = smem;
    float* Kj = Qi + Br * d;
    float* Vj = Kj + Bc * d;
    float* Sij = Vj + Bc * d;
    float* Oi = Sij + Br * Bc;
    float* li = Oi + Br * d;
    float* li_new = li + Br;
    float* mi = li_new + Br;
    float* mi_new = mi + Br;
    float* mij_dash = mi_new + Br;

    for (int j = 0; j < Tc; j++) {
        // load Kj and Vj into SMEM
        // a thread may load multiple elements
        int loads_per_thread = CEIL_DIV(d, Br);
        for (int e = 0; e < loads_per_thread; e++) {
            int idx = e * (Br * Bc) + tx;
            if (idx < Bc * d) {
                int row = idx / d;
                int col = idx % d;

                if (j * Bc + row < N) {
                    Kj[row * d + col] = K[qkv_off + (j * Bc + row) * d + col];
                    Vj[row * d + col] = V[qkv_off + (j * Bc + row) * d + col];
                }
            }
        }
        __syncthreads();  // barrier here for correct Kj and Vj values in inner loop

        for (int i = 0; i < Tr; i++) {
            // load Qi and Oi into smem similar to Kj
            // a thread may load multiple elements
            int loads_per_thread = CEIL_DIV(d, Bc);
            for (int e = 0; e < loads_per_thread; e++) {
                int idx = e * (Br * Bc) + tx;
                if (idx < Br * d) {
                    int row = idx / d;
                    int col = idx % d;
                    if (i * Br + row < N) {
                        Qi[row * d + col] = Q[qkv_off + (i * Br + row) * d + col];
                        Oi[row * d + col] = O[qkv_off + (i * Br + row) * d + col];
                    }
                }
            }

            int s_row = tx / Bc;
            int s_col = tx % Bc;

            if (s_col == 0) {
                mi[s_row] = m[lm_off + (i * Br) + s_row];
                li[s_row] = l[lm_off + (i * Br) + s_row];
            }
            __syncthreads();

            // compute S = Qi * Kj^T where shape of S: (Br, Bc)
            // TODO: reduce shared memory bank conflicts
            float acc = 0.f;
            for (int k = 0; k < d; k++)
                acc += Qi[s_row * d + k] * Kj[s_col * d + k];

            acc *= scale;
            Sij[s_row * Bc + s_col] = acc;

            // rowmax(S) and rowsum(S) (only one thread per row)
            // computes both in a single pass
            if (s_col == 0) {
                float row_m = -INFINITY, row_l = 0.f;
                for (int c = 0; c < Bc; c++) {
                    float val = Sij[s_row * Bc + c];
                    if (val > row_m) {
                        row_m = val;
                    }
                }
                for (int c = 0; c < Bc; c++) {
                    float exp_val = expf(Sij[s_row * Bc + c] - row_m);
                    Sij[s_row * Bc + c] = exp_val;
                    row_l += exp_val;
                }

                mij_dash[s_row] = row_m;
                mi_new[s_row] = max(mi[s_row], row_m);
                li_new[s_row] = expf(mi[s_row] - mi_new[s_row]) * li[s_row] + expf(row_m - mi_new[s_row]) * row_l;
            }
            __syncthreads();

            // compute Sij * Vj and do a roll-forward update to O
            // Sij (Br, Bc) and Vj (Bc, d) and we have Br * Bc threads
            // a thread may compute more than one element's dot product
            for (int col = s_col; col < d; col += Bc) {
                float acc = 0.f;
                for (int c = 0; c < Bc; c++)
                    acc += Sij[s_row * Bc + c] * Vj[c * d + col];

                int global_row = (i * Br) + s_row;
                if (global_row < N) {
                    Oi[s_row * d + col] = (1 / li_new[s_row]) * ((li[s_row] * expf(mi[s_row] - mi_new[s_row]) * Oi[s_row * d + col]) + (expf(mij_dash[s_row] - mi_new[s_row]) * acc));
                    O[qkv_off + global_row * d + col] = Oi[s_row * d + col];
                }
            }

            // update max and norm for next iteration
            m[lm_off + (i * Br) + s_row] = mi_new[s_row];
            l[lm_off + (i * Br) + s_row] = li_new[s_row];
        }
        __syncthreads();
    }
}

// Comment the below function to compile and run this file as executable
torch::Tensor fa_forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    // TODO: determine Bc, Br dynamically
    const int Bc = 16;
    const int Br = 16;

    int B = Q.size(0);
    int nh = Q.size(1);
    int N = Q.size(2);
    int d = Q.size(3);

    int Tc = ceil((float)N / Bc);
    int Tr = ceil((float)N / Br);
    float softmax_scale = 1.0 / sqrt(d);

    // Initialize O, l, m to HBM
    auto O = torch::zeros_like(Q);
    auto l = torch::zeros({B, nh, N});
    auto m = torch::full({B, nh, N}, -INFINITY);
    torch::Device device(torch::kCUDA);
    l = l.to(device);
    m = m.to(device);

    const int smem_size = ((Br * Bc) + (2 * Br * d) + (2 * Bc * d) + (5 * Br)) * sizeof(float);
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, smem_size);

    dim3 grid_size(B, nh);     // batch_size x num_heads
    dim3 block_size(Br * Bc);  // Br * Bc threads per block

    flash_attn_1_kernel<Br, Bc><<<grid_size, block_size, smem_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
        N, d, Tr, Tc, softmax_scale,
        l.data_ptr<float>(), m.data_ptr<float>(), O.data_ptr<float>());
    return O;
}

int main() {
    int batch_size = 16;
    int n_head = 8;
    int seq_len = 512;
    int head_embd = 64;

    int qkv_size = batch_size * n_head * seq_len * head_embd;
    int lm_size = batch_size * n_head * seq_len;

    float *Qh, *Kh, *Vh, *Oh, *lh, *mh;
    Qh = (float*)malloc(qkv_size * sizeof(float));
    Kh = (float*)malloc(qkv_size * sizeof(float));
    Vh = (float*)malloc(qkv_size * sizeof(float));
    Oh = (float*)malloc(qkv_size * sizeof(float));
    lh = (float*)malloc(lm_size * sizeof(float));
    mh = (float*)malloc(lm_size * sizeof(float));

    for (int i = 0; i < qkv_size; i++) {
        Qh[i] = random_normal_clamped(-1, 1);
        Kh[i] = random_normal_clamped(-1, 1);
        Vh[i] = random_normal_clamped(-1, 1);
        Oh[i] = 0.0f;
    }
    for (int i = 0; i < lm_size; i++) {
        lh[i] = 0.0f;
        mh[i] = -INFINITY;
    }

    const int Br = 16, Bc = 16;
    int Tc = ceil((float)seq_len / Bc);
    int Tr = ceil((float)seq_len / Br);
    float softmax_scale = 1.0 / sqrt(head_embd);

    const int smem_size = ((Br * Bc) + (2 * Br * head_embd) + (2 * Bc * head_embd) + (5 * Br)) * sizeof(float);
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, smem_size);

    dim3 grid_dim(batch_size, n_head);  // batch_size x num_heads
    dim3 block_dim(Br * Bc);            // Br * Bc threads per block

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    float *Q, *K, *V, *O, *l, *m;

    hipEventRecord(start);
    CUDA_CHECK(hipMalloc(&Q, qkv_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&K, qkv_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&V, qkv_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&O, qkv_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&l, lm_size * sizeof(float)));
    CUDA_CHECK(hipMalloc(&m, lm_size * sizeof(float)));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> GPU allocation time: %f ms\n", ms);

    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(Q, Qh, qkv_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(K, Kh, qkv_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(V, Vh, qkv_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(O, Oh, qkv_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(l, lh, lm_size * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(m, mh, lm_size * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Host to device transfer time: %f ms\n", ms);

    hipEventRecord(start);
    flash_attn_1_kernel<Br, Bc><<<grid_dim, block_dim, smem_size>>>(
        Q, K, V, seq_len, head_embd, Tr, Tc, softmax_scale,
        l, m, O);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Flash-Attention 1 kernel execution time: %f ms\n", ms);

    hipEventRecord(start);
    CUDA_CHECK(hipMemcpy(Oh, O, qkv_size * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf(">> Device to host transfer time: %f ms\n", ms);

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    printf("\nFirst and Last value in Output:\n");
    printf("%f and %f\n", Oh[0], Oh[qkv_size - 1]);

    hipFree(Q);
    hipFree(K);
    hipFree(V);
    hipFree(O);
    hipFree(l);
    hipFree(m);
    free(Qh);
    free(Kh);
    free(Vh);
    free(Oh);
    free(lh);
    free(mh);

    return 0;
}