#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

int main() {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int M = 2, N = 3, K = 4;
    float *h_A, *h_B, *h_C;
    h_A = (float *)malloc(M * K * sizeof(float));
    h_B = (float *)malloc(K * N * sizeof(float));
    h_C = (float *)malloc(M * N * sizeof(float));

    for (int i = 0; i < M; i++)
        for (int j = 0; j < K; j++)
            h_A[i * K + j] = i + j;

    for (int i = 0; i < K; i++)
        for (int j = 0; j < N; j++)
            h_B[i * N + j] = i + j;

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    const float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                M, N, K, &alpha,
                d_A, M, d_B, K,
                &beta, d_C, M);

    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Matrix A:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < K; j++) {
            printf("%f ", h_A[i * K + j]);
        }
        printf("\n");
    }

    printf("Matrix B:\n");
    for (int i = 0; i < K; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_B[i * N + j]);
        }
        printf("\n");
    }

    printf("Matrix C = A * B:\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", h_C[i + j * M]);
        }
        printf("\n");
    }

    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipblasDestroy(handle);
    return 0;
}
