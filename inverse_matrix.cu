#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <vector>
#include <chrono>
#include <random>
#include <cassert>

#define BLOCK_SIZE 16
#define TOL 1e-4

void matrix_inverse_cpu(float* A, int n) {
    std::vector<float> aug(n * 2 * n, 0);
    
    
    for(int i = 0; i < n; ++i) {
        for(int j = 0; j < n; ++j)
            aug[i * 2 * n + j] = A[i * n + j];
        aug[i * 2 * n + n + i] = 1.0f;
    }
    for(int col = 0; col < n; ++col) {
        
        int max_row = col;
        for(int i = col+1; i < n; ++i)
            if(fabs(aug[i * 2 * n + col]) > fabs(aug[max_row * 2 * n + col]))
                max_row = i;
        
        if(max_row != col)
            for(int j = col; j < 2 * n; ++j)
                std::swap(aug[col * 2 * n + j], aug[max_row * 2 * n + j]);
        
        float pivot = aug[col * 2 * n + col];
        for(int j = col; j < 2 * n; ++j)
            aug[col * 2 * n + j] /= pivot;
        
        for(int i = 0; i < n; ++i) {
            if(i != col) {
                float factor = aug[i * 2 * n + col];
                for(int j = col; j < 2 * n; ++j)
                    aug[i * 2 * n + j] -= factor * aug[col * 2 * n + j];
            }
        }
    }
    
    for(int i = 0; i < n; ++i)
        for(int j = 0; j < n; ++j)
            A[i * n + j] = aug[i * 2 * n + n + j];
}

__global__ void init_aug_kernel(float* A, float* aug, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(row < n && col < 2 * n) {
        if(col < n) { 
            aug[row * 2 * n + col] = A[row * n + col];
        } else {      
            aug[row * 2 * n + col] = (col == row + n) ? 1.0f : 0.0f;
        }
    }
}
__global__ void swap_rows_kernel(float* aug, int n, int col, int max_row) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(j >= 2 * n || col == max_row) return;
    
    float temp = aug[col * 2 * n + j];
    aug[col * 2 * n + j] = aug[max_row * 2 * n + j];
    aug[max_row * 2 * n + j] = temp;
}
__global__ void normalize_kernel(float* aug, int n, int col) {
    int j = blockIdx.x * blockDim.x + threadIdx.x + col;
    if(j >= 2 * n) return;
    float pivot = aug[col * 2 * n + col];
    if(pivot != 0.0f && j >= col)
        aug[col * 2 * n + j] /= pivot;
}
__global__ void eliminate_kernel(float* aug, int n, int col) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y + col;
    
    if(row >= n || j >= 2 * n || row == col) return;
    
    float factor = aug[row * 2 * n + col];
    aug[row * 2 * n + j] -= factor * aug[col * 2 * n + j];
}

void matrix_inverse_gpu(float* d_A, int n) {
    float *d_aug;
    hipMalloc(&d_aug, n * 2 * n * sizeof(float));
    
    dim3 init_block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 init_grid(
        (2 * n + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (n + BLOCK_SIZE - 1) / BLOCK_SIZE
    );
    init_aug_kernel<<<init_grid, init_block>>>(d_A, d_aug, n);
    hipDeviceSynchronize();
    for(int col = 0; col < n; ++col) {
        
        int num_rows = n - col;
        std::vector<float> h_col(num_rows);
        hipMemcpy(&h_col[0],
                   d_aug + col * 2 * n + col,
                   num_rows * sizeof(float),
                   hipMemcpyDeviceToHost);
        
        int max_row = col;
        float max_val = fabs(h_col[0]);
        for (int i = 1; i < num_rows; i++) {
            float val = fabs(h_col[i]);
            if(val > max_val) {
                max_val = val;
                max_row = col + i;
            }
        }
        
        dim3 swap_block(256);
        dim3 swap_grid((2 * n + 255) / 256);
        swap_rows_kernel<<<swap_grid, swap_block>>>(d_aug, n, col, max_row);
        hipDeviceSynchronize();
        
        normalize_kernel<<<(2 * n - col + 255) / 256, 256>>>(d_aug, n, col);
        hipDeviceSynchronize();
        
        
        dim3 elim_block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 elim_grid(
            (n + BLOCK_SIZE - 1) / BLOCK_SIZE, 
            ((2 * n - col) + BLOCK_SIZE - 1) / BLOCK_SIZE
        );
        eliminate_kernel<<<elim_grid, elim_block>>>(d_aug, n, col);
        hipDeviceSynchronize();
    }
    
    
    
    hipMemcpy2D(d_A, n * sizeof(float),
                 d_aug + n, 2 * n * sizeof(float),
                 n * sizeof(float), n,
                 hipMemcpyDeviceToDevice);
    
    hipFree(d_aug);
}


void test_matrix_inverse(int n) {
    std::vector<float> h_A(n * n);
    std::vector<float> h_A_inv_cpu(n * n);
    std::vector<float> h_A_inv_gpu(n * n);
    
    
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(1.0f, 10.0f);
    
    for(int i = 0; i < n; ++i) {
        float sum = 0.0f;
        for(int j = 0; j < n; ++j) {
            h_A[i * n + j] = dist(gen);
            sum += fabs(h_A[i * n + j]);
        }
        h_A[i * n + i] = sum + 1.0f; 
    }
    
    
    h_A_inv_cpu = h_A;
    auto start_cpu = std::chrono::high_resolution_clock::now();
    matrix_inverse_cpu(h_A_inv_cpu.data(), n);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    float cpu_time = std::chrono::duration<float>(end_cpu - start_cpu).count();
    
    float *d_A;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMemcpy(d_A, h_A.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    matrix_inverse_gpu(d_A, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    hipMemcpy(h_A_inv_gpu.data(), d_A, n * n * sizeof(float), hipMemcpyDeviceToHost);
    
    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);
    gpu_time /= 1000.0f; 
    
    
    float max_error = 0.0f;
    for(int i = 0; i < n * n; ++i)
        max_error = fmaxf(max_error, fabsf(h_A_inv_cpu[i] - h_A_inv_gpu[i]));
    
    std::cout << "Matrix Size: " << n << "x" << n << "\n";
    std::cout << "Max Error: " << max_error << "\n";
    std::cout << "CPU Time: " << cpu_time << " s\n";
    std::cout << "GPU Time: " << gpu_time << " s\n";
    std::cout << "Speedup: " << cpu_time / gpu_time << "x\n\n";
    
    hipFree(d_A);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
int main() {
    test_matrix_inverse(256);
    test_matrix_inverse(512);
    test_matrix_inverse(1024);
    return 0;
}
