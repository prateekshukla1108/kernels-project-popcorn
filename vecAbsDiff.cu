#include "hip/hip_runtime.h"
#include "helpers.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>

__global__
void vectorAbsDiffKernel(float *d_Output, const float *d_A, const float *d_B, int size){
    // automatic variable
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size){
        float diff = d_A[i] - d_B[i];
        float absDiff;
        // if negative, multiply with -1, else keep same. 
        if (diff >= 0){
            absDiff = diff;
        } else {
            absDiff = -diff;
        }
        d_Output[i] = absDiff;
    }
}

int main(){
    int size = 10;
    // memory size in size of floats
    size_t memSize = size * sizeof(float);

    std::vector<float> A(size), B(size), Output(size);

    // initializing with sample data
    for (int i=0; i < size; ++i){
        A[i] = i + 1.0f;
        B[i] = size - i;
    }

    std::cout << "values in vector A:";
    for (float val : A) 
        std::cout << val << " ";
    std::cout << "values in vector B:";
    for (float val : B) 
        std::cout << val << " ";

    // allocate device memory. "stub"
    float *d_A, *d_B, *d_Output;
    // creating a variable to check for cuda errors
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&d_A, memSize);
    if (cudaStatus != hipSuccess){
        std::cerr << "hipMalloc d_A failed! " << hipGetErrorString(cudaStatus) << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc((void**)&d_B, memSize);
    if (cudaStatus != hipSuccess){
        std::cerr << "hipMalloc d_B failed! " << hipGetErrorString(cudaStatus) << std::endl;
        // freeing already allocated d_A because we crashin
        hipFree(d_A);
        return 1;
    }
    
    cudaStatus = hipMalloc((void**)&d_Output, memSize);
    if (cudaStatus != hipSuccess){
        std::cerr << "hipMalloc d_Output failed! " << hipGetErrorString(cudaStatus) << std::endl;
        // freeing d_A and d_B
        hipFree(d_B);
        hipFree(d_A);
        return 1;
    }

    cudaStatus = hipMemcpy(d_A, A.data(), memSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess){
        std::cerr << "copy to d_A failed! " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_Output);
        return 1;
    }
    

    cudaStatus = hipMemcpy(d_B, B.data(), memSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess){
        std::cerr << "copy to d_B failed! " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_Output);
        return 1;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1)/threadsPerBlock;
    // creating cuda events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // recording 'start' event at 0
    hipEventRecord(start, 0);
    // launching the kernel
    vectorAbsDiffKernel<<<blocksPerGrid, threadsPerBlock>>>(d_Output, d_A, d_B, size);

    CHECK_KERNEL_ERROR();

    // recording 'stop' event
    hipEventRecord(stop, 0);
    // wait for stop event to finish to be recorded and gpu to finish
    hipEventSynchronize(stop);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess){
        std::cerr << "kernel launch failed! " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_Output);
        return 1;
    }

    // wait for kernel execution to complete before hipMemcpyDeviceToHost
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess){
        std::cerr << "hipDeviceSynchronize failed! " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_Output);
    }

    // copy output
    cudaStatus = hipMemcpy(Output.data(), d_Output, memSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess){
        std::cerr << "cudaMemCpy d_Output to Output failed! " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_Output);
        return 1;
    }

    std::cout << "absolute difference: ";
    for (float val: Output) std::cout << val << " ";
    std::cout << std::endl;

    std::cout << "verification (host): ";
    for (int i = 0; i < size; ++i){
        std::cout << std::abs(A[i] - B[i]) << " ";
    }
    std::cout << std::endl;

    cudaStatus = hipFree(d_A);
    if (cudaStatus != hipSuccess){
        std::cerr << "hipFree d_A failed! " << hipGetErrorString(cudaStatus) << std::endl;
    }

    cudaStatus = hipFree(d_B);
    if (cudaStatus != hipSuccess){
        std::cerr << "hipFree d_B failed! " << hipGetErrorString(cudaStatus) << std::endl;
    }

    cudaStatus = hipFree(d_Output);
    if (cudaStatus != hipSuccess){
        std::cerr << "hipFree d_Output failed! " << hipGetErrorString(cudaStatus) << std::endl;
    }

    // calculating execution time in milliseconds
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);
    // milliseconds = milliseconds/100;

    printf("\n Kernel Execution Time: %.3f milliseconds\n", milliseconds);

    return 0;

}