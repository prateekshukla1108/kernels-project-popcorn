#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <cstdlib>   
#include <iostream>  // For std::cout
using namespace std; // For cout

#define TileWidth 2
__global__ void tiledMatMul(float *d_A, float *d_B, float *d_C, int Width){

    //Shared memory declaration
    __shared__ float Mds[TileWidth][TileWidth];
    __shared__ float Nds[TileWidth][TileWidth];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;

    // Compute row and column indices
    int row = by * TileWidth + ty;
    int col = bx * TileWidth + tx;

    // Boundary Check
    if (row < Width && col < Width){

    // Init sum
    float Pvalue = 0.0;

    // number of tiles needed
    int nTiles = Width/TileWidth;

    for (int ph=0; ph < nTiles; ++ph){

        Mds[ty][tx] = d_A[row * Width + (ph * TileWidth + tx)];
        Nds[ty][tx] = d_B[(ph * TileWidth + ty) * Width + col];
        __syncthreads(); // wait for all threads to fetch their items

        for (int k=0; k < TileWidth; k++){
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads(); // wait for all threads to finish matmul
    }
        d_C[row * Width + col] = Pvalue; //assign
    }
}

int main(){

    int Width = 4; // ensure divisible by TileWidth
    // int TileWidth = 2;

    int Size = Width * Width * sizeof(float); // dont forget float

    //Init and Allocate Host ptrs and memory
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C; 
    hipHostMalloc(&h_A, Size, hipHostMallocDefault);
    hipHostMalloc(&h_B, Size, hipHostMallocDefault);
    hipHostMalloc(&h_C, Size, hipHostMallocDefault);
    hipMalloc(&d_A, Size);
    hipMalloc(&d_B, Size);
    hipMalloc(&d_C, Size);

    // Init host matrices
    for(int i = 0; i < Width*Width; ++i) {
        h_A[i] = rand() % 10 ;  // Random integers between 0 and 9
        h_B[i] = rand() % 10 ;  // Random integers between 0 and 9
    }
        // Print Matrix A
    cout << "\nMatrix A:\n";
    for(int i = 0; i < Width; ++i) {
        for(int j = 0; j < Width; ++j) {
            cout << static_cast<int>(h_A[i*Width + j]) << " ";
        }
        cout << endl;
    }

    // Print Matrix B
    cout << "\nMatrix B:\n";
    for(int i = 0; i < Width; ++i) {
        for(int j = 0; j < Width; ++j) {
            cout << static_cast<int>(h_B[i*Width + j]) << " ";
        }
        cout << endl;
    }
    
    // Copy from Host to Device
    hipMemcpy(d_A, h_A, Size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, Size, hipMemcpyHostToDevice);

    // Define grid and block dimensions and launch kernel
    dim3 blockSize(TileWidth, TileWidth);
    dim3 gridSize((Width + TileWidth - 1) / TileWidth,
    (Width + TileWidth - 1) / TileWidth);
    
    tiledMatMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, Width);

    // error checking
    hipError_t err;
    err = hipGetLastError(); // `cudaGetLastError` will return the error from above.
    if (err != hipSuccess)
    {
      printf("Error: %s\n", hipGetErrorString(err));
    }

    //copy back the result
    hipMemcpy(h_C, d_C, Size, hipMemcpyDeviceToHost);

       // Print Result Matrix C
    cout << "\nResult Matrix C:\n";
    for(int i = 0; i < Width; ++i) {
        for(int j = 0; j < Width; ++j) {
            cout << static_cast<int>(h_C[i*Width + j]) << " ";
        }
        cout << endl;
    }

    // Free memory
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;

}
