
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloCUDA()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}