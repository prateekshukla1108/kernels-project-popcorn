#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "utils.cuh"

/*
Naive Sgemv kernel

- Each thread calculates one element of the output vector
- The row index is calculated using block index and thread index
- Uses linearized indexing
- Memory accesses are not coalesced
*/
__global__ void naive_sgemv_kernel(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M) {
        float sum = 0.0f;
        for (int col = 0; col < N; col++) {
            sum += matd[row * N + col] * vecd[col];
        }
        resd[row] = sum;
    }
}

/*
Runs the naive Sgemv kernel.
*/
float run_kernel_naive_sgemv(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N, float THEORETICAL_MAX_GFLOPS, float THEORETICAL_MAX_MEMORY_BANDWIDTH) {
    dim3 block_size(1024);
    dim3 grid_size(CEIL_DIV(M, block_size.x));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.f;

    CUDA_CHECK(hipEventRecord(start));
    naive_sgemv_kernel<<<grid_size, block_size>>>(matd, vecd, resd, M, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    printf("------- Naive sgmev kernel ---------\n");
    print_kernel_essentials(M, N, ms, THEORETICAL_MAX_GFLOPS, THEORETICAL_MAX_MEMORY_BANDWIDTH);
    printf("---------------------------\n");

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return ms;
}
