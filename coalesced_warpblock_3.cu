#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "utils.cuh"

/*
Coalesced Warp Block Sgemv kernel

- Each block is assigned to a row of the matrix A
- Each block calculates one output element of y
- The columns are accessed in coalesced manner by threads
- Performs warp level + block level sum reduction
*/
__global__ void coalesced_warpblock_sgmev_kernel(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N) {
    extern __shared__ float smem[];

    int bid = blockIdx.x;
    if (bid >= M) return;

    int tid = threadIdx.x;
    // each thread calculates its own partial output
    float partial_sum = 0.f;
    for (int col = tid; col < N; col += blockDim.x) {
        partial_sum += matd[bid * N + col] * vecd[col];
    }

    // block level sum reduction
    // only first thread reads the first location in shared memory
    // only first thread writes the output to global memory
    blockReduceSum(partial_sum, smem, tid, blockDim.x);
    if (tid == 0) {
        float sum = smem[0];
        resd[bid] = sum;
    }
}

/*
Runs the coalesced warp sgemv kernel.
*/
float run_kernel_coalesced_warpblock_sgmev(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N, float THEORETICAL_MAX_GFLOPS, float THEORETICAL_MAX_MEMORY_BANDWIDTH) {
    int NUM_THREADS = 64;
    int warp_size = 32;

    dim3 block_size(NUM_THREADS);
    dim3 grid_size(M);
    size_t shared_mem_size = CEIL_DIV(block_size.x, warp_size) * sizeof(float);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.f;

    CUDA_CHECK(hipEventRecord(start));
    coalesced_warpblock_sgmev_kernel<<<grid_size, block_size, shared_mem_size>>>(matd, vecd, resd, M, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    printf("------- Coalesced warp-block sgmev kernel ---------\n");
    print_kernel_essentials(M, N, ms, THEORETICAL_MAX_GFLOPS, THEORETICAL_MAX_MEMORY_BANDWIDTH);
    printf("---------------------------\n");

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return ms;
}
