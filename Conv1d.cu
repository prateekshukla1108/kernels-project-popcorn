#include <iostream>
#include <hip/hip_runtime.h>

#define N 16
#define F 3

__global__ void Conv1D(float *input, float *filter, float *output, int input_size, int filter_size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < input_size - filter_size + 1){
        float result = 0.0f;
        for (int j = 0; j < filter_size; j++){
            result += input[i + j] * filter[j];
        }
        output[i] = result;
    }
}

int main()
{
    float input[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    float filter[F] = {0.25, 0.5, 0.25}; // Example filter (simple smoothing filter)
    float output[N - F + 1]; // Output size is input size minus filter size + 1

    float *d_input, *d_filter, *d_output;

    hipMalloc((void **)&d_input, N * sizeof(float));
    hipMalloc((void **)&d_filter, N * sizeof(float));
    hipMalloc((void **)&d_output, (N - F + 1) * sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, F * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 16;
    int grid_size = (N - F + 1 + block_size -1 ) / block_size;
    Conv1D<<<grid_size, block_size>>>(d_input, d_filter, d_output, N, F);

    hipMemcpy(output, d_output, (N - F +1 )* sizeof(float), hipMemcpyDeviceToHost);
    // Print the result
    std::cout << "Input signal: ";
    for (int i = 0; i < N; ++i) {
        std::cout << input[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Filtered output: ";
    for (int i = 0; i < N - F + 1; ++i) {
        std::cout << output[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_input);
    hipFree(d_filter);
    hipFree(d_output);

    return 0;
}
