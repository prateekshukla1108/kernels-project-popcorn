#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel function
__global__ void multiplyArrays(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        c[idx] = a[idx] * b[idx];
    }
}

int main() {
    int n = 100;
    size_t size = n * sizeof(int);

    int *h_a = new int[n];
    int *h_b = new int[n];
    int *h_c = new int[n];

    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    multiplyArrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++) {
        std::cout << h_a[i] << " * " << h_b[i] << " = " << h_c[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
