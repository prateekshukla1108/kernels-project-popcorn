#include "hip/hip_runtime.h"
/*
A simple implementation of CUDA kernel for softmax function
*/

#include <stdio.h>
#include <stdlib.h>
#include "utils.h"

/*
The below kernel parallelises the computation of softmax function.
- each thread is responsible for calculating the the softmax for a row.
- the below is a naive implementation. (we can optimize this by 
*/ 
__global__ void softMaxNaive(float *logits, float *results, int N, int K){
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N){
        float norm = 0.0f;
        float n_max = -INFINITY;
        
        // pass for calculating the maximum value in the vector.
        for(int i=0; i<K; i++){
            if(n_max<logits[row*K+i]){
                n_max = logits[row*K+i];
            }
        }
        // a pass for calculating 'norm'. i.e, summation of e^V_i
        for(int i=0; i<K; i++){
            norm += exp(logits[row*K+i]-n_max);
        }
        // a pass again for calcualting the softmax for each element.
        for(int i=0; i<K; i++){
            results[row*K+i] = exp(logits[row*K+i]-n_max)/norm;
        }
    }
}

void printMatrix(float *M, int N, int K);
struct timer t;

int main(){
    int N = 1000000;  // number of rows
    int K = 512;  // number of cols (elements per row)

    size_t bytes = N * K * sizeof(float); // num of bytes required in memory

    float *logits, *results;

    logits = (float*)malloc(bytes);
    results = (float*)malloc(bytes);

    // initialise values
    for(int i=0; i<N*K; i++){
        logits[i] = i+1;
    }

    // printf("Logits matrix: \n");
    // printMatrix(logits, N, K);

    float *logits_d, *results_d;

    hipMalloc((void**)&logits_d, bytes);
    hipMalloc((void**)&results_d, bytes);

    hipMemcpy(logits_d, logits, bytes, hipMemcpyHostToDevice);

    // kernel launch
    dim3 THREADS(32, 1, 1);
    dim3 BLOCKS((K+ THREADS.x- 1)/THREADS.x, 1, 1);

    start_timer(&t);
    softMaxNaive<<<BLOCKS, THREADS>>>(logits_d, results_d, N, K);
    hipDeviceSynchronize();
    stop_timer(&t);

    hipMemcpy(results, results_d, bytes, hipMemcpyDeviceToHost);

    // printf("probablities from softmax: \n");
    // printMatrix(results, N, K);

    printf("Time taken to compute softmax: %f seconds\n", time_diff(&t));

    free(logits);
    free(results);
    hipFree(logits_d);
    hipFree(results_d);
    return 0;
}

void printMatrix(float *M, int N, int K){
    for(int i=0;i<N; i++){
        for(int j=0; j<K; j++){
            printf("%f ", M[i*K+j]);
        }printf("\n");
    }printf("\n");
}