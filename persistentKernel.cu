#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>

#define IMAGESIZE 1024  // 32 x 32 x 1
#define NUM_IMAGES 1024 // Total number of images
#define THREADS_PER_BLOCK 256

__device__ float dummyKernel(const float *image)
{
    float sum = 0.0f;
    for (int i = 0; i < IMAGESIZE; i++)
    {
        sum += image[i];
    }
    return sum;
}

__global__ void persistentKernel(const float *ImageData, float *output, const int *taskQueue,
                                 int numTask, volatile int *QueueHead,
                                 volatile bool *doneFlag)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    while (!(*doneFlag))
    {
        int taskIndex = atomicAdd((int *)QueueHead, 1);
        if (taskIndex < numTask)
        {
            int imageIndex = taskQueue[taskIndex];
            int imageOffset = imageIndex * IMAGESIZE;
            float result = dummyKernel(&ImageData[imageOffset]);
            output[imageIndex] = result;
        }
        else
        {
            __nanosleep(100);
            if (atomicAdd((int *)QueueHead, 0) >= numTask)
            {
                *doneFlag = true;
            }
        }
    }
}

__global__ void normalKernel(const float *ImageData, float *output)
{
    int imageIndex = blockDim.x * blockIdx.x + threadIdx.x;
    if (imageIndex < NUM_IMAGES)
    {
        int imageOffset = imageIndex * IMAGESIZE;
        float result = dummyKernel(&ImageData[imageOffset]);
        output[imageIndex] = result;
    }
}

int main()
{
    size_t imageDataSize = NUM_IMAGES * IMAGESIZE * sizeof(float);
    float *h_ImageData = new float[NUM_IMAGES * IMAGESIZE];
    float *h_output = new float[NUM_IMAGES];
    int *h_taskQueue = new int[NUM_IMAGES];

    for (int i = 0; i < NUM_IMAGES; i++)
    {
        for (int j = 0; j < IMAGESIZE; j++)
        {
            h_ImageData[i * IMAGESIZE + j] = static_cast<float>(rand()) / RAND_MAX;
        }
        h_taskQueue[i] = i;
    }

    // Device allocations.
    float *d_ImageData, *d_output;
    int *d_taskQueue;
    int *d_QueueHead;
    bool *d_doneFlag;
    hipMalloc(&d_ImageData, imageDataSize);
    hipMalloc(&d_output, NUM_IMAGES * sizeof(float));
    hipMalloc(&d_taskQueue, NUM_IMAGES * sizeof(int));
    hipMalloc(&d_QueueHead, sizeof(int));
    hipMalloc(&d_doneFlag, sizeof(bool));

    hipMemcpy(d_ImageData, h_ImageData, imageDataSize, hipMemcpyHostToDevice);
    hipMemcpy(d_taskQueue, h_taskQueue, NUM_IMAGES * sizeof(int), hipMemcpyHostToDevice);
    int zero = 0;
    bool falseFlag = false;
    hipMemcpy(d_QueueHead, &zero, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_doneFlag, &falseFlag, sizeof(bool), hipMemcpyHostToDevice);

    int blocks = (THREADS_PER_BLOCK > NUM_IMAGES ? 1 : (NUM_IMAGES + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

    hipEvent_t startPersistent, stopPersistent;
    hipEventCreate(&startPersistent);
    hipEventCreate(&stopPersistent);
    hipEventRecord(startPersistent);

    persistentKernel<<<blocks, THREADS_PER_BLOCK>>>(d_ImageData, d_output, d_taskQueue, NUM_IMAGES, d_QueueHead, d_doneFlag);
    hipEventRecord(stopPersistent);
    hipEventSynchronize(stopPersistent);
    float persistentTime;
    hipEventElapsedTime(&persistentTime, startPersistent, stopPersistent);

    hipMemcpy(h_output, d_output, NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Persistent Kernel Execution Time: " << persistentTime << " ms" << std::endl;

    hipMemset(d_output, 0, NUM_IMAGES * sizeof(float));

    hipEvent_t startNormal, stopNormal;
    hipEventCreate(&startNormal);
    hipEventCreate(&stopNormal);
    hipEventRecord(startNormal);

    normalKernel<<<blocks, THREADS_PER_BLOCK>>>(d_ImageData, d_output);
    hipEventRecord(stopNormal);
    hipEventSynchronize(stopNormal);
    float normalTime;
    hipEventElapsedTime(&normalTime, startNormal, stopNormal);

    hipMemcpy(h_output, d_output, NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Normal Kernel Execution Time: " << normalTime << " ms" << std::endl;

    hipFree(d_ImageData);
    hipFree(d_output);
    hipFree(d_taskQueue);
    hipFree(d_QueueHead);
    hipFree(d_doneFlag);
    delete[] h_ImageData;
    delete[] h_output;
    delete[] h_taskQueue;

    hipEventDestroy(startPersistent);
    hipEventDestroy(stopPersistent);
    hipEventDestroy(startNormal);
    hipEventDestroy(stopNormal);

    return 0;
}
