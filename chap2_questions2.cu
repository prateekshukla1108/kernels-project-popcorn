#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include "common.h"


__global__ void question1(const float *A, const float *B, float *C, int N) {
    // block_id * block_size + thread_id
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Cuda launches 
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}


// CUDA kernel for vector addition
__global__ void question2(const float *A, const float *B, float *C, int N) {
    // Each thread processes two adjacent elements
    int i = (blockIdx.x*blockDim.x + threadIdx.x) * 2;

    // Cuda launches 
    if (i < N) {
        C[i] = A[i] + B[i];
    }

    if (i + 1 < N) {
        C[i + 1] = A[i + 1] + B[i + 1];
    }
}

__global__ void question3(const float *A, const float *B, float *C, int N) {
    // Each thread block will process 2*blockDim
    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Cuda launches 
    if (i < N) {
        C[i] = A[i] + B[i];
    }

    if (i + blockDim.x < N) {
        C[i + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
    }
}

void vectorAddCPU(const float *A, const float *B, float *C, int N) {
    for (int i = 0; i < N; i++) {
        C[i] = A[i] + B[i];
    }
}


int main() {
    // 1M elements
    int N = 1 << 20;
    size_t size = N * sizeof(float);

    // Allocate memory on the host
    float *A_host = (float*)malloc(size);
    float *B_host = (float*)malloc(size);
    float *C_mat_cpu = (float*)malloc(size);
    float *C_mat_gpu = (float*)malloc(size);

    // Initialize input vectors
    initializeVectors(A_host, B_host, N);

    // Measure CPU time for vector addition
    double cpuTime = measureExecutionTime([&]() {
        vectorAddCPU(A_host, B_host, C_mat_cpu, N);
    });
    std::cout << "CPU execution time: " << cpuTime << " ms" << std::endl;

    // Allocate memory on the device
    float *A_device, *B_device, *C_device;
    hipMalloc((void**)&A_device, size);
    hipMalloc((void**)&B_device, size);
    hipMalloc((void**)&C_device, size);

    // Copy data from host to device
    hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice);
    hipMemcpy(B_device, B_host, size, hipMemcpyHostToDevice);

    // Define execution configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Measure GPU time for vector addition
    double gpuTime = measureExecutionTime([&]() {
        question3<<<blocksPerGrid, threadsPerBlock>>>(A_device, B_device, C_device, N);
        hipDeviceSynchronize();
    });
    std::cout << "GPU execution time: " << gpuTime << " ms" << std::endl;

    // Copy result back to host
    hipMemcpy(C_mat_gpu, C_device, size, hipMemcpyDeviceToHost);

    // Compare CPU and GPU results
    bool success = compareResults(C_mat_cpu, C_mat_gpu, N);
    std::cout << (success ? "CPU and GPU results match!" : "Results mismatch!") << std::endl;

    // Free device memory
    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);

    // Free host memory
    free(A_host);
    free(B_host);
    free(C_mat_cpu);
    free(C_mat_gpu);

    return 0;

}
