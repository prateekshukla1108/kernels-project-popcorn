#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK_CUDA(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// optimized ReLU Kernel
__global__ void reluKernel(const float* __restrict__ input, float* __restrict__ output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = fmaxf(0.0f, input[idx]); // ReLU operation
    }
}

void reluCUDA(float* h_input, float* h_output, int N) {
    float *d_input, *d_output;
    size_t size = N * sizeof(float);

    CHECK_CUDA(hipMalloc((void**)&d_input, size));
    CHECK_CUDA(hipMalloc((void**)&d_output, size));

    CHECK_CUDA(hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice));

    int blockSize = 256; 
    int gridSize = (N + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    CHECK_CUDA(hipEventRecord(start));

    reluKernel<<<gridSize, blockSize>>>(d_input, d_output, N);

    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float time = 0;
    CHECK_CUDA(hipEventElapsedTime(&time, start, stop));
    std::cout << "Kernel execution time: " << time << " ms" << std::endl;

    CHECK_CUDA(hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));

    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
}

int main() {
    const int N = 1 << 20; // 1M elements
    float *h_input = new float[N];
    float *h_output = new float[N];

    for (int i = 0; i < N; i++) {
        h_input[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f; 
    }

    reluCUDA(h_input, h_output, N);

    for (int i = 0; i < N; i++) {
        if (h_output[i] != fmaxf(0.0f, h_input[i])) {
            std::cerr << "Error at index " << i << std::endl;
            break;
        }
    }

    delete[] h_input;
    delete[] h_output;

    return 0;
}
// Kernel execution time: 0.383968 ms