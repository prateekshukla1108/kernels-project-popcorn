#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define n 10

int main()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int j;
    float *x, *y;
    x = (float *)malloc(sizeof(float) * n);
    y = (float *)malloc(sizeof(float) * n);
    for (j = 0; j < n; ++j)
    {
        x[j] = (float)1;
        y[j] = (float)1;
    }

    printf("\nx:\n");
    for (j = 0; j < n; ++j)
    {
        printf("%f ", x[j]);
    }

    printf("\ny:\n");
    for (j = 0; j < n; ++j)
    {
        printf("%f ", y[j]);
    }

    float *d_x, *d_y;
    cudaStat = hipMalloc((void **)&d_x, n * sizeof(float));
    cudaStat = hipMalloc((void **)&d_y, n * sizeof(float));

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
    stat = hipblasSetVector(n, sizeof(float), y, 1, d_y, 1);
    float a = 3.0;

    float result;
    stat = hipblasSdot(handle, n, d_x, 1, d_y, 1, &result);
    printf("\ndot product x . y : \n ");
    printf (" %7.0f \n " , result );


    hipFree(d_y);
    hipFree(d_x);
    hipblasDestroy(handle);
    free(x);
    free(y);
    return 0;
}