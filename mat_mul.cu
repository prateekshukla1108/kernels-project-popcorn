
#include <hip/hip_runtime.h>
#include <iostream>

/*

[[1,2],   .  [[1,3],    =   [[5, 11],
 [3,4]]       [2,4]]         [11, 25]]

*/

__global__ void matmul(int* X, int* Y, int* Z, int width){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int temp = 0;
    for(int k = 0; k<width; k++){
        temp += X[tx * width + k] * Y[width * k + ty];
    }
    Z[tx*width+ty] = temp;
}

int main( void ) {

    int N = 2;
    int size = N*N*sizeof(int);

    int h_X[4] = {1, 2, 3, 4};  
    int h_Y[4] = {1, 3, 2, 4};
    int h_Z[N*N];

    int *d_X, *d_Y, *d_Z;

    hipMalloc((void**)&d_X, size); // allocate memory
    hipMalloc((void**)&d_Y, size);
    hipMalloc((void**)&d_Z, size);

    hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, size, hipMemcpyHostToDevice);
    hipMemcpy(d_Z, h_Z, size, hipMemcpyHostToDevice);

    dim3 grid_size(1);
    dim3 block_size(N, N);
    
    matmul<<<grid_size, block_size>>>(d_X, d_Y, d_Z, N);

    hipMemcpy(h_Z, d_Z, size, hipMemcpyDeviceToHost);

    hipFree(d_X);   // free memory
    hipFree(d_Y);
    hipFree(d_Z);

    std::cout << "Result matrix Z:" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_Z[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}