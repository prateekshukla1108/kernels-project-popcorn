#include "bfs.h"
#include "bfs_kernel.cu"

/*
We call the bfs_kernel many time from the host, as a first approximiation, I think this is better than trying to fuse everything in the kernel since it can lead
to warp divergence. However, I believe there is a better way to do this for sure, like using private and globa queus which I'll do when I need this algorithm in an application of mine.

*/

void bfs_gpu(int source, int num_vertices, int num_edges, int* h_edges, int* h_dest, int* h_labels) {
    int *d_edges, *d_dest, *d_labels, *d_done;
    
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_edges, (num_vertices + 1) * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_dest, num_edges * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_labels, num_vertices * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_done, sizeof(int)));
    
    CHECK_CUDA_ERROR(hipMemset(d_labels, -1, num_vertices * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemcpy(d_edges, h_edges, (num_vertices + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_dest, h_dest, num_edges * sizeof(int), hipMemcpyHostToDevice));
    
    int initial_level = 0;
    CHECK_CUDA_ERROR(hipMemcpy(d_labels + source, &initial_level, sizeof(int), hipMemcpyHostToDevice));
    
    int level = 0;
    int h_done;
    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid = (num_vertices + threadsPerBlock - 1) / threadsPerBlock;
    
    do {
        h_done = 1;
        CHECK_CUDA_ERROR(hipMemcpy(d_done, &h_done, sizeof(int), hipMemcpyHostToDevice));
        
        bfs_kernel<<<blocksPerGrid, threadsPerBlock>>>(level, num_vertices, d_edges, d_dest, d_labels, d_done);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        CHECK_CUDA_ERROR(hipMemcpy(&h_done, d_done, sizeof(int), hipMemcpyDeviceToHost));
        level++;
    } while (!h_done && level < num_vertices);
    
    CHECK_CUDA_ERROR(hipMemcpy(h_labels, d_labels, num_vertices * sizeof(int), hipMemcpyDeviceToHost));
    
    CHECK_CUDA_ERROR(hipFree(d_edges));
    CHECK_CUDA_ERROR(hipFree(d_dest));
    CHECK_CUDA_ERROR(hipFree(d_labels));
    CHECK_CUDA_ERROR(hipFree(d_done));
}