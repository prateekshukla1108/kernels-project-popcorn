#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <random>

// Kernel function for SwiGLU
__global__ void swiglu_kernel(float* out, const float* x, const float* W1, const float* W2, int batch_size, int hidden_dim, int output_dim) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int o = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < batch_size && o < output_dim) {
        float xW1 = 0.0f;
        float xW2 = 0.0f;
        
        for (int i = 0; i < hidden_dim; i++) {
            xW1 += x[b * hidden_dim + i] * W1[o + i * output_dim];
            xW2 += x[b * hidden_dim + i] * W2[o + i * output_dim];
        }
        
        float sigmoid_val = 1.0f / (1.0f + expf(-xW1));
        float result = xW1 * sigmoid_val * xW2;
        
        if (b == 0 && o == 0) {  // Print debug info for first element
            printf("GPU Debug: xW1=%f, xW2=%f, sigmoid_val=%f, result=%f\n", 
                   xW1, xW2, sigmoid_val, result);
        }
        
        out[b * output_dim + o] = result;
    }
}

void swiglu_forward(float* out, const float* x, const float* W1, const float* W2, int batch_size, int hidden_dim, int output_dim) {
    // Allocate memory on GPU
    float *d_x, *d_W1, *d_W2, *d_out;
    hipMalloc((void**)&d_x, batch_size * hidden_dim * sizeof(float));
    hipMalloc((void**)&d_W1, hidden_dim * output_dim * sizeof(float));
    hipMalloc((void**)&d_W2, hidden_dim * output_dim * sizeof(float));
    hipMalloc((void**)&d_out, batch_size * output_dim * sizeof(float));
    
    // Copy data to GPU
    hipMemcpy(d_x, x, batch_size * hidden_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W1, W1, hidden_dim * output_dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W2, W2, hidden_dim * output_dim * sizeof(float), hipMemcpyHostToDevice);
    
    // Define CUDA kernel launch parameters
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((batch_size + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (output_dim + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Launch kernel
    swiglu_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_x, d_W1, d_W2, batch_size, hidden_dim, output_dim);
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
    }
    
    // Copy result back to CPU
    hipMemcpy(out, d_out, batch_size * output_dim * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free GPU memory
    hipFree(d_x);
    hipFree(d_W1);
    hipFree(d_W2);
    hipFree(d_out);
}

int main() {
    int batch_size = 32;
    int hidden_dim = 128;
    int output_dim = 64;
    
    // Allocate memory
    float *x = new float[batch_size * hidden_dim];
    float *W1 = new float[hidden_dim * output_dim];
    float *W2 = new float[hidden_dim * output_dim];
    float *out = new float[batch_size * output_dim];
    
    // Initialize random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 1.0f);
    
    // Initialize input data with random values between 0 and 1
    for (int i = 0; i < batch_size * hidden_dim; i++) {
        x[i] = dis(gen);
    }
    for (int i = 0; i < hidden_dim * output_dim; i++) {
        W1[i] = dis(gen);
        W2[i] = dis(gen);
    }
    
    // Manual CPU calculation for first element (for verification)
    float manual_xW1 = 0.0f;
    float manual_xW2 = 0.0f;
    for (int i = 0; i < hidden_dim; i++) {
        manual_xW1 += x[i] * W1[i * output_dim];
        manual_xW2 += x[i] * W2[i * output_dim];
    }
    std::cout << "CPU Manual calculation for first element:" << std::endl;
    std::cout << "xW1: " << manual_xW1 << std::endl;
    std::cout << "xW2: " << manual_xW2 << std::endl;
    float manual_sigmoid = 1.0f / (1.0f + exp(-manual_xW1));
    float manual_result = manual_xW1 * manual_sigmoid * manual_xW2;
    std::cout << "Expected result: " << manual_result << std::endl;
    
    // Compute SwiGLU
    swiglu_forward(out, x, W1, W2, batch_size, hidden_dim, output_dim);
    
    // Print some input values
    std::cout << "\nFirst 10 input values:" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << "x[" << i << "]: " << x[i] << std::endl;
    }
    
    std::cout << "\nFirst 10 W1 values:" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << "W1[" << i << "]: " << W1[i] << std::endl;
    }
    
    std::cout << "\nFirst 10 W2 values:" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << "W2[" << i << "]: " << W2[i] << std::endl;
    }
    
    // Print output values
    std::cout << "\nFirst 10 output values:" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << "out[" << i << "]: " << out[i] << std::endl;
    }
    
    // Free memory
    delete[] x;
    delete[] W1;
    delete[] W2;
    delete[] out;
    
    return 0;
}
