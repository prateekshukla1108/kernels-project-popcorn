#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    // Initialize CUDA
    hipInit(0);

    // Create a CUDA context
    hipCtx_t context;
    hipCtxCreate(&context, 0, 0);

    // Load the PTX file with printf support
    hipModule_t module;
    hipJitOption options[] = { hipJitOptionLogVerbose };
    void* optionValues[] = { (void*)1 }; // Enable verbose logging (includes printf support)
    hipModuleLoadDataEx(&module, "hello.ptx", 1, options, optionValues);

    // Get the kernel function from the PTX module
    hipFunction_t kernel;
    hipModuleGetFunction(&kernel, module, "my_kernel");

    // Launch the kernel
    void *args[] = {};  // No arguments for this kernel
    hipModuleLaunchKernel(kernel, 1, 1, 1, 1, 1, 1, 0, 0, args, 0);

    // Synchronize to ensure the kernel completes
    hipCtxSynchronize();

    // Clean up
    hipModuleUnload(module);
    hipCtxDestroy(context);

    return 0;
}