#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>


#define n 6

int main(){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int j;
    float *x;
    x = (float*)malloc(sizeof(float)*n);
    for(j = 0 ; j<n ;++j){
        x[j] = (float)j;
    }

    printf("x:\n");
    for(j = 0 ; j<n ;++j){
        printf("%f\n",x[j]);
    }

    float *d_x;
    cudaStat = hipMalloc((void**)&d_x,n*sizeof(float));

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n,sizeof(float),x,1,d_x,1);
    
    int result;

    stat = hipblasIsamax(handle,n,d_x,1,&result);
    printf("max: %f\n",fabs(x[result-1]));

    stat = hipblasIsamin(handle,n,d_x,1,&result);
    printf("min: %f\n",fabs(x[result-1]));

    hipFree(d_x);
    hipblasDestroy(handle);
    free(x);
    return 0;
}