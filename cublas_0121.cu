#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "utils.cuh"

/*
CuBLAS matrix vector multiplication for the baseline scores.
We simply run the Sgemv function that cuBLAS provides.
*/
float run_kernel_cublas_sgemv(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N, float THEORETICAL_MAX_GFLOPS, float THEORETICAL_MAX_MEMORY_BANDWIDTH) {
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.0f;

    // create cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Sgemv: y = (alpha * A * x) + (beta * y)
    float alpha = 1.0f, beta = 0.0f;
    hipEventRecord(start);
    hipblasSgemv(handle, HIPBLAS_OP_T, N, M, &alpha, matd, N, vecd, 1, &beta, resd, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    printf("------- cuBLAS sgmev kernel ---------\n");
    print_kernel_essentials(M, N, ms, THEORETICAL_MAX_GFLOPS, THEORETICAL_MAX_MEMORY_BANDWIDTH);
    printf("---------------------------\n");

    hipblasDestroy(handle);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return ms;
}
