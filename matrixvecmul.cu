
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorMatrixM(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
      float sum=0.0f;
      for (int j = 0; j < N; j++) {
         sum += A[i*N+j]*B[j];
      }
      C[i]=sum;
}}

int main() {
    const int N = 10;
    float *A, *B, *C;
    A = (float *)malloc( N*N* sizeof(float));
    B = (float *)malloc(N*sizeof(float));
    C = (float *)malloc(N*sizeof(float));


    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            A[i * N + j] = 1.0f;
        }
        B[i] = 2.0f;
        C[i] = 0.0f;
    }

    float *d_a, *d_b,*d_c;
    hipMalloc(&d_a,N*N*sizeof(float));
    hipMalloc(&d_b,N*sizeof(float));
    hipMalloc(&d_c,N*sizeof(float));
    hipMemcpy(d_a,A,N*N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,B,N*sizeof(float),hipMemcpyHostToDevice);
    int blocksize=256;
    int gridsize = (N + blocksize - 1) / blocksize;
    vectorMatrixM<<<gridsize,blocksize>>>(d_a,d_b,d_c,N);

    hipDeviceSynchronize();
    hipMemcpy(C,d_c,N*sizeof(float),hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%f ", B[i]);

    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}