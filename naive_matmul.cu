#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define CHECK_CUDA_CALL(err)                                                \
    {                                                                       \
        if (err != hipSuccess)                                             \
        {                                                                   \
            fprintf(stderr, "CUDA error in file %s at line %d: %s\n",       \
                    __FILE__, __LINE__, hipGetErrorString(err));           \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    }


__global__
void matrix_multiplication_naive(float *A, float* B, float *result, int rows_result, int col_result, int inner_dim){
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < rows_result && col < col_result){
        // printf ("Executing for  (%d, %d)\n", row, col);
        int idx = row * col_result + col;

        float dot_prod = 0.0;
        for (int i=0; i < inner_dim; ++i){
            dot_prod += A[row * inner_dim + i] * B[i * col_result+ col];
        }
        result[idx] = dot_prod;
    }
}


float* matrix_multiplication(float *h_a, float *h_b, int row_a, int col_a, int row_b, int col_b){
    float *d_a, *d_b, *d_result;
    int size_a = sizeof(float) * row_a * col_a;
    int size_b = sizeof(float) * row_b * col_b;
    int size_result = sizeof(float) * row_a * col_b;
    float *h_result = new float[size_result];

    //Allocate device memory
    hipError_t err = hipMalloc((void**) &d_a, size_a);
    CHECK_CUDA_CALL(err);
    err = hipMalloc((void**) &d_b, size_b);
    CHECK_CUDA_CALL(err);
    err = hipMalloc((void**) &d_result, size_result);
    CHECK_CUDA_CALL(err);
    
    //copy matrices to device
    err = hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    CHECK_CUDA_CALL(err);
    err = hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    CHECK_CUDA_CALL(err);
    
    
    int thread_x = 16;
    int thread_y = 16;
    dim3 block_dims(thread_x, thread_y, 1);
    int blocks_x = (col_b + thread_x - 1)/thread_x;
    int blocks_y = (row_a + thread_y - 1)/thread_y;
    dim3 grid_dims(blocks_x, blocks_y, 1);
    

    matrix_multiplication_naive<<<grid_dims, block_dims>>>(d_a, d_b, d_result, row_a, col_b, col_a);

    // copy result to host
    err = hipMemcpy(h_result, d_result, size_result, hipMemcpyDeviceToHost);
    CHECK_CUDA_CALL(err);

    //free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return h_result;
}


void test_matrix_multiplication(){
    cout << "Running Test 1:\n";
    float *A = new float[6];
    float *B = new float[8];

    fill_n(A, 6, 1.0f);
    fill_n(B, 8, 1.0f);

    float *C = matrix_multiplication(A, B, 3, 2, 2, 4);

    for (int i=0; i< 3; ++i){
        for (int j=0; j<4; ++j){
            cout << C[i * 4 + j] << " ";
        }
        cout << "\n";
    }


    cout << "\nRunning test 2:\n";

    A = new float[50*50];
    B = new float[2500];

    for (int i=0; i < 50; ++i){
        A[i*50 + i] = 1;
        B[i*50 + i] = 1;
    }
    C = matrix_multiplication(A, B, 50, 50, 50, 50);
    for (int i=0; i< 50; ++i){
        for (int j=0; j<50; ++j){
            cout << C[i * 50 + j] << " ";
        }
        cout << "\n";
    }
}


int main(){
    test_matrix_multiplication();
}