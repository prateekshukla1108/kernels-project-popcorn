#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 4  

__global__ void softmax(float *input, float *output, int n) {
    __shared__ float sum;

    int idx = threadIdx.x;
    if (idx < n) {
        output[idx] = expf(input[idx]);
    }

    __syncthreads();
    if (idx == 0) {
        sum = 0.0f;
        for (int i = 0; i < n; i++) {
            sum += output[i];
        }
    }

    __syncthreads();

    if (idx < n) {
        output[idx] /= sum;
    }
}

int main() {
    float h_input[N] = {1.0, 2.0, 3.0, 4.0};
    float h_output[N];

    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    softmax<<<1, N>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Softmax Output:\n";
    for (int i = 0; i < N; i++) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}

