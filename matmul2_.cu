#include<iostream>
#include<hip/hip_runtime.h>


// mat mul kernel
__global__ void matmulkernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0;
        for (int i = 0; i < N; ++i) {
            value += A[row * N + i] * B[i * N + col]; 
        }
        C[row * N + col] = value;
    }
}

void matmul(float* h_A, float* h_B, float* h_C, int N) {
    int size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(32, 32);
    dim3 dimGrid((N + 32 - 1) / 32, (N + 32 - 1) / 32);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matmulkernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Kernel Execution Time: " << time << "ms" << std::endl;

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int N = 1024;
    int size = N * N * sizeof(float);

    float *h_A, *h_B, *h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    matmul(h_C, h_A, h_B, N);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
// Kernel Execution Time: 3.74483ms