#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define n 10

int main()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int j;
    float *x, *y;
    x = (float *)malloc(sizeof(float) * n);
    y = (float *)malloc(sizeof(float) * n);

    for (j = 0; j < n; ++j)
    {
        x[j] = (float)j;
        y[j] = (float)j + 1;
    }

    printf("\nx:\n");
    for (j = 0; j < n; ++j)
    {
        printf("%f ", x[j]);
    }

    printf("\ny:\n");
    for (j = 0; j < n; ++j)
    {
        printf("%f ", y[j]);
    }

    float *d_x, *d_y;
    cudaStat = hipMalloc((void **)&d_x, n * sizeof(float));
    cudaStat = hipMalloc((void **)&d_y, n * sizeof(float));

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
    stat = hipblasSetVector(n, sizeof(float), y, 1, d_y, 1);
    float a = 3.0;

    stat = hipblasSaxpy(handle, n, &a, d_x, 1, d_y, 1);
    stat = hipblasGetVector(n, sizeof(float), d_y, 1, y, 1);

    printf("\nNew y:\n");
    for (j = 0; j < n; ++j)
    {
        printf("%f ", y[j]);
    }
    hipFree(d_y);
    hipFree(d_x);
    hipblasDestroy(handle);
    free(x);
    free(y);
    return 0;
}