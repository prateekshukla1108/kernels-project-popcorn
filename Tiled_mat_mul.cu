#include "hip/hip_runtime.h"
﻿#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

const int N = 1 << 10;
const int SHMEM_SIZE = 1 << 10;

__global__ void matrixMul(const int* a, const int* b, int* c) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int s_a[SHMEM_SIZE];
    __shared__ int s_b[SHMEM_SIZE];

    int tmp = 0;

    for (int i = 0; i < N; i += blockDim.x) {
  
        s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * N + i + threadIdx.x];
        s_b[threadIdx.y * blockDim.x + threadIdx.x] =
            b[i * N + threadIdx.y * N + col];

        __syncthreads();

        for (int j = 0; j < blockDim.x; j++) {
            tmp +=
                s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
        }

        __syncthreads();
    }

    c[row * N + col] = tmp;
}

void verify_result(vector<int>& a, vector<int>& b, vector<int>& c) {

    for (int i = 0; i < N; i++) {

        for (int j = 0; j < N; j++) {
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                tmp += a[i * N + k] * b[k * N + j];
            }
            assert(tmp == c[i * N + j]);
        }
    }
}

int main() {

    size_t bytes = N * N * sizeof(int);

    vector<int> h_a(N * N);
    vector<int> h_b(N * N);
    vector<int> h_c(N * N);

    generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
    generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = 32;
    int BLOCKS = N / THREADS;

    dim3 threads(THREADS, THREADS);
    dim3 blocks(BLOCKS, BLOCKS);

    matrixMul << <blocks, threads >> > (d_a, d_b, d_c);

    hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    verify_result(h_a, h_b, h_c);

     printf("The Result is %d\n", d_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
