
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define SEQ_LEN 64
#define DIM 64
#define TILE_SIZE 16

__global__ void flashAttentionKernel(const float* Q, const float* K, const float* V, float* output, int seqLen, int dim) {
    
    __shared__ float Q_tile[TILE_SIZE][TILE_SIZE];
    __shared__ float K_tile[TILE_SIZE][TILE_SIZE];
    __shared__ float V_tile[TILE_SIZE][TILE_SIZE];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    
    float local_sum = 0.0f;
    float local_max = -INFINITY;
    
    float out_val = 0.0f;
    
    for (int t = 0; t < seqLen / TILE_SIZE; t++) {
        
        Q_tile[ty][tx] = (row < seqLen && t * TILE_SIZE + tx < dim) ? Q[row * dim + t * TILE_SIZE + tx] : 0.0f;
        
        K_tile[ty][tx] = (col < seqLen && t * TILE_SIZE + ty < dim) ? K[col * dim + t * TILE_SIZE + ty] : 0.0f;
        
        V_tile[ty][tx] = (t * TILE_SIZE + ty < seqLen && tx < dim) ? V[(t * TILE_SIZE + ty) * dim + tx] : 0.0f;
        __syncthreads();
        
        float score = 0.0f;
        for (int i = 0; i < TILE_SIZE; i++) {
            score += Q_tile[ty][i] * K_tile[i][tx]; 
        }
        score /= sqrtf(dim);
        
        local_max = fmaxf(local_max, score);
        local_sum += expf(score - local_max);
        
        for (int i = 0; i < TILE_SIZE; i++) {
            out_val += expf(score - local_max) * V_tile[i][tx]; 
        }
        __syncthreads();
    }
    
    out_val /= local_sum;
    
    if (row < seqLen && col < dim) {
        output[row * dim + col] = out_val;
    }
}


int main() {
    int seqLen = SEQ_LEN;
    int dim = DIM;
    size_t matrixSize = seqLen * dim * sizeof(float);
    
    float *h_Q = (float*)malloc(matrixSize);
    float *h_K = (float*)malloc(matrixSize);
    float *h_V = (float*)malloc(matrixSize);
    float *h_output = (float*)malloc(matrixSize);
    
    for (int i = 0; i < seqLen * dim; i++) {
        h_Q[i] = static_cast<float>(rand()) / RAND_MAX;
        h_K[i] = static_cast<float>(rand()) / RAND_MAX;
        h_V[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    
    float *d_Q, *d_K, *d_V, *d_output;
    hipMalloc(&d_Q, matrixSize);
    hipMalloc(&d_K, matrixSize);
    hipMalloc(&d_V, matrixSize);
    hipMalloc(&d_output, matrixSize);
    
    hipMemcpy(d_Q, h_Q, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, matrixSize, hipMemcpyHostToDevice);
    
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((dim + TILE_SIZE - 1) / TILE_SIZE, (seqLen + TILE_SIZE - 1) / TILE_SIZE);
    
    flashAttentionKernel<<<blocksPerGrid, threadsPerBlock>>>(d_Q, d_K, d_V, d_output, seqLen, dim);
    
    hipMemcpy(h_output, d_output, matrixSize, hipMemcpyDeviceToHost);
    
    printf("Output matrix element at (0, 0): %f\n", h_output[0]);
    
    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_output);
    
    free(h_Q);
    free(h_K);
    free(h_V);
    free(h_output);
    return 0;
}
