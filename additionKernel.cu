#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>

template <typename T>
__global__ void addKernel(T* input, int arraySize) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arraySize) {
        input[idx] += 10;
    }
}

void addition(torch::Tensor& input, int arraySize) {
    int threads_per_block = 256;
    int blocks = (arraySize + threads_per_block - 1) / threads_per_block;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "arrayAddition", [&]() {
        addKernel<<<blocks, threads_per_block>>>(input.data_ptr<scalar_t>(), arraySize);
    });
    hipDeviceSynchronize();
    
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        TORCH_CHECK(false, "CUDA error: ", hipGetErrorString(err));
    }
}   