// matrix multiplication using cuBLAS

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <hiprand/hiprand.h>

#define N 1024


int main(){
    
    size_t matrix_size = N*N*sizeof(float);
    float alpha = 1.0f;
    float beta = 0.0f;

    // allocate host memory
    float *h_A = (float*)malloc(matrix_size);
    float *h_B = (float*)malloc(matrix_size);
    float *h_C = (float*)malloc(matrix_size);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, matrix_size);
    hipMalloc((void**)&d_B, matrix_size);
    hipMalloc((void**)&d_C, matrix_size);

    
    for (int i = 0; i<N*N; i++){
        h_A[i] = (i % N) + 1;   // 1, 2, 3, ..., N, columnwise
        h_B[i] = (i / N) + 1;  // 1,2,3...N rowwise
    }

    

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetMatrix(N, N, sizeof(float), h_A, N, d_A, N);
    hipblasSetMatrix(N, N, sizeof(float), h_B, N, d_B, N);
    // cublasSetMatrix(N, N, sizeof(float), h_C, N, d_C, N);
    hipMemset(d_C, 0, matrix_size);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N,N,N, &alpha, d_A, N, d_B, N, &beta, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);  // Ensure kernel execution is finished

    // Compute elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms\n";


    
    hipblasGetMatrix(N, N, sizeof(float), d_C, N, h_C, N);
    hipblasDestroy(handle);

    printf("Successful");
    printf("Sample result C[0][0] = %f\n", h_C[0]);

    // free memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;

}