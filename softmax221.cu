
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define VECTOR_SIZE 1024
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

__global__ void softmaxKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = exp(input[idx]);
    }
}

__global__ void normalizeKernel(float* output, int size, float sum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = output[idx] / sum;
    }
}

int main() {
    int size = VECTOR_SIZE;
    size_t vectorSize = size * sizeof(float);

    float *h_input = (float*)malloc(vectorSize);
    float *h_output = (float*)malloc(vectorSize);

    srand(1234); 
    for (int i = 0; i < size; i++) {
        h_input[i] = static_cast<float>(rand()) / RAND_MAX; 
    }

    
    printf("Input vector (first 5 elements):\n");
    for (int i = 0; i < 5; i++) {
        printf("%f ", h_input[i]);
    }
    printf("\n");

    float *d_input, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, vectorSize));
    CUDA_CHECK(hipMalloc(&d_output, vectorSize));
    CUDA_CHECK(hipMemcpy(d_input, h_input, vectorSize, hipMemcpyHostToDevice));

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    softmaxKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    float h_sum = 0.0f;
    CUDA_CHECK(hipMemcpy(h_output, d_output, vectorSize, hipMemcpyDeviceToHost));
    for (int i = 0; i < size; i++) {
        h_sum += h_output[i];
    }

    printf("Sum of exponentials: %f\n", h_sum);

    normalizeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, size, h_sum);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(h_output, d_output, vectorSize, hipMemcpyDeviceToHost));

    printf("Softmax probabilities for the first 5 elements:\n");
    for (int i = 0; i < 5; i++) {
        printf("%f ", h_output[i]);
    }
    printf("\n");

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    free(h_input);
    free(h_output);

    return 0;
}

