#include <iostream>
#include <hip/hip_runtime.h>

#include <chrono>


#define N 512

__global__ void device_add(int *d_x, int *d_y, int *d_z){
      int idx = blockIdx.x;
      d_z[idx] = d_x[idx] + d_y[idx]; 
}
void host_add(int *a, int *b, int *c){
    for(int i=0;i<N;i++){
        c[i] = a[i] + b[i];
    }
}
void fill_array(int *a){
    for(int i=0;i<N;i++){
        a[i] = i; 
    }
}
void print_out(int *a, int *b, int *c){
    for(int i=0;i<N;i++){
        printf("[%d] + [%d] = %d\n", a[i], b[i], c[i]);
    }
}
int main()  {
    int *a,*b,*c;
    int size = N * sizeof(int);
    a = (int *)malloc(size);
    fill_array(a);
    b = (int *)malloc(size);
    fill_array(b);
    c = (int *)malloc(size);
    fill_array(c);
    auto start_host = std::chrono::high_resolution_clock::now();

    host_add(a,b,c);
        auto stop_host = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> host_time = stop_host - start_host;

    print_out(a,b,c);

    int *d_x;
    int *d_y;
    int *d_z;
    hipMalloc(&d_x,N * sizeof(int));
    hipMalloc(&d_y,N * sizeof(int));
    hipMalloc(&d_z,N * sizeof(int));
    
    hipMemcpy(d_x, a, N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_y, b, N * sizeof(int),hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start recording time
    hipEventRecord(start, 0);
    device_add<<<N,1>>>(d_x,d_y,d_z);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(c, d_z, N*sizeof(int),hipMemcpyDeviceToHost );

    print_out(a,b,c);

     printf("GPU Kernel execution time: %f ms\n", milliseconds);
     printf("CPU execution time: %f ms\n", host_time.count());

    free(a); free(b); free(c);

    hipFree(d_x); hipFree(d_y); hipFree(d_z);
    return 0;

}

