
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void add (int a, int b, int *c){
    *c = a + b;
}

int main (void){
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));

    add <<<1,1>>> (2,7, dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    cout << "2 + 7 = " << c << endl;

    hipFree(dev_c);
    return 0;
}