#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../helpers/cuda_helpers.h"

#define WIDTH 2

// Kernel to calculate P = M * N where M and N are square matrices
__global__ void simpleMatrixMulKernel(float* M, float* N, float* P, int width){
    int Row = blockDim.x * blockIdx.x + threadIdx.x;
    int Col = blockDim.y * blockIdx.y + threadIdx.y;

    if ((Row < width) && (Col < width)){
        float Pvalue = 0;
        for (int i = 0; i < width; i++){
            // Since here we access element of M and one element of N
            // and we apply one addition and one multiplication.
            // compute to global-memory-access ratio is 1. 
            // But we need to increase this ratio!
            Pvalue += M[Row * width + i] * N[i * width + Col];
        }
        P[Row * width + Col] = Pvalue;
    }
}


int main() {
    float h_M[WIDTH * WIDTH] = {1, 2, 3, 4};
    float h_N[WIDTH * WIDTH] = {5, 6, 7, 8};
    float h_P[WIDTH * WIDTH];

    float *d_M, *d_N, *d_P;

    hipMalloc((void**)&d_M, WIDTH * WIDTH * sizeof(float));
    hipMalloc((void**)&d_N, WIDTH * WIDTH * sizeof(float));
    hipMalloc((void**)&d_P, WIDTH * WIDTH * sizeof(float));

    hipMemcpy(d_M, h_M, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, WIDTH * WIDTH * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(WIDTH, WIDTH);
    dim3 dimGrid(1, 1);

    simpleMatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P, WIDTH);

    hipMemcpy(h_P, d_P, WIDTH * WIDTH * sizeof(float), hipMemcpyDeviceToHost);

    printArray(h_M, WIDTH, WIDTH, "matrix M");
    printArray(h_N, WIDTH, WIDTH, "matrix N");
    printArray(h_P, WIDTH, WIDTH, "matrix P");

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}