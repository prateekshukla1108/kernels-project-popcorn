﻿#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void atomicAddKernel(int* sum) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	atomicAdd(sum, tid); //safe addition to memory
 }

int main() {
	int h_sum = 0;  //host value
	int *d_sum;    // device

	//Alocate mem on gpu

	hipMalloc((void**)&d_sum, sizeof(int));
	hipMemcpy(d_sum, &h_sum, sizeof(int), hipMemcpyHostToDevice);

	atomicAddKernel<<<1, 256>>>(d_sum); //launch kernel with 256 thread and 1 block

	hipMemcpy(&h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

	printf("The value after atomicAdd is %d\n", h_sum);

	hipFree(d_sum); //free memory

	return 0;
}
