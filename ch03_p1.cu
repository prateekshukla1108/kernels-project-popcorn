#include "hip/hip_runtime.h"
/*
matrix addition

1. Write a kernel that has each thread producing one output matrix element. Fill in the execution configuration parameters for the design.
2. Write a kernel that has each thread producing one output matrix row. Fill in the execution configuration parameters for the design.
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void mykernel1(int *a, int *b, int *o, int n){

    // get the absolute co-ordinate of thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < n && col < n){
        int idx = row * n + col;
        o[idx] = a[idx] + b[idx];
    }

}

__global__
void mykernel2(int *a, int *b, int *o, int n){

    // get the absolute co-ordinate of thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < n && col < n){
        int idx = row * n + col;
        o[idx] = a[idx] + b[idx];
    }

}

int main(){
    int n = 5;
    int *h_a, *h_b, *h_o;
    int *d_a, *d_b, *d_o;

    // allocate host memory
    h_a = (int *)malloc(n * n * sizeof(int));
    h_b = (int *)malloc(n * n * sizeof(int));
    h_o = (int *)malloc(n * n * sizeof(int));

    // initialize the variable
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            h_a[i * n + j] = i * n + j;
            h_b[i * n + j] = i * n + j;
        }
    }

    // Allocate device memory
    hipMalloc(&d_a, n * n * sizeof(int));
    hipMalloc(&d_b, n * n * sizeof(int));
    hipMalloc(&d_o, n * n * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_a, h_a, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * n * sizeof(int), hipMemcpyHostToDevice);

    // define blocksize and grid size
    dim3 blockDim(2, 2);
    dim3 gridDim(3, 3);

    // launch the kernel
    mykernel<<<gridDim, blockDim>>>(d_a, d_b, d_o, n);

    // synchronize
    hipDeviceSynchronize();

    // Copy data from device to host
    hipMemcpy(h_o, d_o, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // verify the result
    for (int i = 0; i < n; i ++ ){
        for (int j = 0; j < n; j ++){
            printf("h_o[%d][%d] = %d\n", i, j, h_o[i * n + j]);
        }
    }

    // release the memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_o);
    free(h_a);
    free(h_b);
    free(h_o);
}
