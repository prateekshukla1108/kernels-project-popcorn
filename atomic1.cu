#include <hip/hip_runtime.h>
#define N 32  
#include <stdio.h>
__device__ int lane_id() {
    return threadIdx.x & 31;
}

__device__ int atomicAggInc(int *ptr) {
    int mask = __match_any_sync(__activemask(), (unsigned long long)ptr);
    int leader = __ffs(mask) - 1;  
    int res;
    if (lane_id() == leader)       
        res = atomicAdd(ptr, __popc(mask));
    res = __shfl_sync(mask, res, leader); 
    return res + __popc(mask & ((1 << lane_id()) - 1)); 
}

__global__ void test_atomicAggInc(int *d_ptr, int *d_results) {
    int old_val = atomicAggInc(d_ptr);
    d_results[threadIdx.x] = old_val;  
}

int main() {
    int *d_ptr, *d_results;
    int h_ptr = 0;   
    int h_results[N];

    hipMalloc(&d_ptr, sizeof(int));
    hipMalloc(&d_results, N * sizeof(int));

    hipMemcpy(d_ptr, &h_ptr, sizeof(int), hipMemcpyHostToDevice);

    test_atomicAggInc<<<1, N>>>(d_ptr, d_results);

    hipMemcpy(&h_ptr, d_ptr, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_results, d_results, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Final value of ptr: %d\n", h_ptr);
    printf("Old values returned by each thread:\n");
    for (int i = 0; i < N; i++) {
        printf("Thread %2d -> %d\n", i, h_results[i]);
    }

    hipFree(d_ptr);
    hipFree(d_results);

    return 0;
}
