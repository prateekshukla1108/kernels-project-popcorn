﻿#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <assert.h>
#include <stdlib.h>


__global__ void vectorAdd(int* a, int* b, int* c, int n){
	//global thread index (tid)
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
if (tid < n) {
	c[tid] = a[tid] + b[tid];
	}	
}
//initialize vector of size n to int b/w 0-99

void matrix_init(int* a, int n) {
	for (int i = 0; i < n; i++) {
		a[i] = rand() % 100;
	}
}

void error_check(int* a, int* b, int* c, int n) {
	for (int i = 0; i < n; i++) {
		assert(c[i] == a[i] + b[i]);
	}
}

int main() {
//vector of size of 2^16 (65536 elements)
	int n = 1 << 16;
	//Host vector pointers
		int* h_a, * h_b, * h_c;
		//device vector pointer
		int* d_a, * d_b, * d_c;
	//Alocation size for all vectors
	size_t bytes = sizeof(int) * n;

	//Allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	//Allocate device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	//initialize vectors a and b with random values b/w 0 -99
	matrix_init(h_a, n);
	matrix_init(h_b, n);

	//copy data from 

	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	//threadblock size

	int NUM_THREADS = 256;

		//Grid SIze

	int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

	//Launch Kernel on default stream w/o shmem
	vectorAdd <<<NUM_BLOCKS, NUM_THREADS>>> (d_a, d_b, d_c, n);

	//copy sum vector from device to host 
	
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	//check result for error 
	error_check(h_a, h_b, h_c, n);

	printf("Result is %d\n", d_c);

	hipFree(h_a);
	hipFree(h_b);
	hipFree(h_c);
		
	return 0;
}
