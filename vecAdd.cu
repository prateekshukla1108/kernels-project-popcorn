#include <iostream>
#include <hip/hip_runtime.h>

// define the  add kernel 
__global__ void vecAddKernel(float* A, float* B, float* C, int n)
{

int i = blockIdx.x * blockDim.x + threadIdx.x;

if (i<n)
{
    C[i] = A[i] + B[i];
}
}

// wraper function to call the function correctly
void vecAdd(float* A, float* B, float* C, int n){
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    vecAddKernel<<<gridSize, blockSize>>>(A, B, C, n);
}

int main(){

// number of elements in vector
// const int n = 5;
int n;
std::cout << "Enter number of elements for vectors " << std::endl;
std::cin >> n;
// size of vector in bytes
int size = n *sizeof(float);

// init and allocate host input vectors
float *h_A, *h_B, *h_C;
hipHostMalloc(&h_A, size, hipHostMallocDefault);
hipHostMalloc(&h_B, size, hipHostMallocDefault);
hipHostMalloc(&h_C, size, hipHostMallocDefault);

// get vectors from user
std::cout << "Enter " << n << " values for vector A (space-separated): ";
for(int i = 0; i < n; i++) {
     std::cin >> h_A[i];
}

std::cout << "Enter " << n << " values for vector B (space-separated): ";
for(int i = 0; i < n; i++) {
    std::cin >> h_B[i];
}

//init and allocate device memory
float *d_A, *d_B, *d_C;
hipMalloc(&d_A, size);
hipMalloc(&d_B, size);
hipMalloc(&d_C, size);

// copy from host to device
hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

// kernel call
vecAdd(d_A, d_B, d_C, n);

//copy back to host
hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

for(int i = 0; i < n; i++) {
  std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
}

// free memory
hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

hipHostFree(h_A);
hipHostFree(h_B);
hipHostFree(h_C);
return 0;
}
