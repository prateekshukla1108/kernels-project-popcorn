#include <iostream>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32

__global__ void matmulKernel(float *A, float *B, float *C, int dim)
{
    int i, j;       // i and j indexes
    float temp = 0; // temp value

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float ASharedT[BLOCK_SIZE][BLOCK_SIZE]; // we allocate memory for shared
    __shared__ float BSharedT[BLOCK_SIZE][BLOCK_SIZE]; // we allocate memory fro shared

    for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++)
    {
        j = tileNUM * BLOCK_SIZE + threadIdx.x;
        i = tileNUM * BLOCK_SIZE + threadIdx.y;

        ASharedT[threadIdx.y][threadIdx.x] = A[i * dim + j];
        BSharedT[threadIdx.y][threadIdx.x] = B[i * dim + j];

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; k++)
        {
            temp += ASharedT[threadIdx.y][k] * BSharedT[k][threadIdx.x];
        }

        __syncthreads();
    }
    C[row * dim + col] = temp;
}

int main()
{
    int N = 1024;
    float *Acpu, *Bcpu, *Ccpu;
    float *Agpu, *Bgpu, *Cgpu;

    Acpu = (float *)malloc(N * N * sizeof(float));
    Bcpu = (float *)malloc(N * N * sizeof(float));
    Ccpu = (float *)malloc(N * N * sizeof(float));

    for (int i = 0; i < N * N; i++)
    {
        Acpu[i] = sin(i);
        Bcpu[i] = cos(i);
    }

    size_t vectorSize = N * N * sizeof(float);

    hipMalloc((void **)&Agpu, vectorSize);
    hipMalloc((void **)&Bgpu, vectorSize);
    hipMalloc((void **)&Cgpu, vectorSize);
    hipMemcpy(Agpu, Acpu, vectorSize, hipMemcpyHostToDevice);
    hipMemcpy(Bgpu, Bcpu, vectorSize, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(N / BLOCK_SIZE, N / BLOCK_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    matmulKernel<<<gridDim, blockDim>>>(Agpu, Bgpu, Cgpu, N);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float et;
    hipEventElapsedTime(&et, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(Ccpu, Cgpu, vectorSize, hipMemcpyDeviceToHost);

    printf("GPU time= %f ms\n", et);

    free(Acpu);
    free(Bcpu);
    free(Ccpu);
    hipFree(Agpu);
    hipFree(Bgpu);
    hipFree(Cgpu);

    return 0;
}