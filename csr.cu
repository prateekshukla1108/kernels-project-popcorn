#include <iostream>
#include <hip/hip_runtime.h>

// spmv csr kernel
__global__ void spmvCSRKernel(int num_rows, const float* values, const int* col_indices, const int* row_ptr, const float* x, float* y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot_product = 0.0f;
        int row_start = row_ptr[row];
        int row_end = row_ptr[row + 1];
        for (int j = row_start; j < row_end; j++) {
            dot_product += values[j] * x[col_indices[j]];
        }
        y[row] = dot_product;
    }
}

void spmvCSR(int num_rows, const float* values, const int* col_indices, const int* row_ptr, const float* x, float* y) {
    float *d_values, *d_x, *d_y;
    int *d_col_indices, *d_row_ptr;

    hipMalloc((void**)&d_values, sizeof(float) * row_ptr[num_rows]);
    hipMalloc((void**)&d_col_indices, sizeof(int) * row_ptr[num_rows]);
    hipMalloc((void**)&d_row_ptr, sizeof(int) * (num_rows + 1));
    hipMalloc((void**)&d_x, sizeof(float) * num_rows);
    hipMalloc((void**)&d_y, sizeof(float) * num_rows);

    hipMemcpy(d_values, values, sizeof(float) * row_ptr[num_rows], hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, col_indices, sizeof(int) * row_ptr[num_rows], hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, sizeof(int) * (num_rows + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, sizeof(float) * num_rows, hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks_per_grid = (num_rows + threads_per_block - 1) / threads_per_block;
    spmvCSRKernel<<<blocks_per_grid, threads_per_block>>>(num_rows, d_values, d_col_indices, d_row_ptr, d_x, d_y);

    hipMemcpy(y, d_y, sizeof(float) * num_rows, hipMemcpyDeviceToHost);

    hipFree(d_values);
    hipFree(d_col_indices);
    hipFree(d_row_ptr);
    hipFree(d_x);
    hipFree(d_y);
}

int main() {
    int num_rows = 1000000; // no of rows(n * n)
    int nnz = 5000000; // no of non-zero elements

    float* values = new float[nnz];
    int* col_indices = new int[nnz];
    int* row_ptr = new int[num_rows + 1];
    float* x = new float[num_rows];
    float* y = new float[num_rows];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    spmvCSR(num_rows, values, col_indices, row_ptr, x, y);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    std::cout << "Kernel execution time: " << time << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    delete[] values;
    delete[] col_indices;
    delete[] row_ptr;
    delete[] x;
    delete[] y;

    return 0;
}
// Kernel execution time: 8.58669 ms