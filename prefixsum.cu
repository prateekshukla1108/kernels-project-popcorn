#include <stdio.h>
#include <hip/hip_runtime.h>


// Naïve prefix sum kernel (inclusive scan)
__global__ void naive_prefix_sum(int *d_in, int *d_out, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < n) {
        int sum = 0;
        for (int i = 0; i <= tid; i++) {
            sum += d_in[i];
        }
        d_out[tid] = sum;
    }
}

int main() {
    const int N = 1024;
    int h_in[N], h_out[N];

    
    for (int i = 0; i < N; i++) {
        h_in[i] = 1;  
    }

    int *d_in, *d_out;
    hipMalloc((void**)&d_in, N * sizeof(int));
    hipMalloc((void**)&d_out, N * sizeof(int));

    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    naive_prefix_sum<<<blocksPerGrid, threadsPerBlock>>>(d_in, d_out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time = 0;
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);


    printf("Kernel execution time: %f ms\n", time);

    hipFree(d_in);
    hipFree(d_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
// Kernel execution time: 55.084927 ms