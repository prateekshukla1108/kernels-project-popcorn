
/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

#define BLOCK_DIM 1024

__global__ void get_sum(float *data, float *result, int nx)
{
    int data_id = blockIdx.y;
    int t = threadIdx.x;
    int i = data_id * nx + 2 * blockIdx.x * BLOCK_DIM + t;
    __shared__ float data_s[BLOCK_DIM];
    data_s[t] = 0.0f;
    __syncthreads();
    if (i < (1 + data_id) * nx)
        data_s[t] = data[i];
    if ((i + BLOCK_DIM) < (1 + data_id) * nx)
        data_s[t] += data[i + BLOCK_DIM];
    for (int stride = BLOCK_DIM / 2; stride >= 1; stride /= 2)
    {
        __syncthreads();
        if (t < stride && (t + stride) < BLOCK_DIM)
        {
            data_s[t] = data_s[t] + data_s[t + stride];
        }
    }
    if (t == 0)
    {
        atomicAdd(&result[data_id], data_s[0]);
    }
}

__global__ void corr(int ny, int nx, const float *data, float *result, float *sums)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (j > i || i >= ny || j >= ny)
        return;
    if (i == j)
    {
        result[i + j * ny] = 1.0f;
        return;
    }
    float avg_i = sums[i] / nx;
    float avg_j = sums[j] / nx;
    float sum_ij = 0.0f;
    float sum_i = 0.0f;
    float sum_j = 0.0f;

    for (int k = 0; k < nx; k++)
    {
        float x = data[k + i * nx];
        float y = data[k + j * nx];
        sum_ij += (x - avg_i) * (y - avg_j);
        sum_i += (x - avg_i) * (x- avg_i);
        sum_j += (y - avg_j) * (y - avg_j);
    }
    result[i + j * ny] = sum_ij / sqrt(sum_i * sum_j);
    result[j + i * ny] = sum_ij / sqrt(sum_i * sum_j);
}

void correlate(int ny, int nx, const float *data, float *result)
{
    float *d_data, *d_sums, *d_result;
    gpuErrchk(hipMalloc((void **)&d_data, nx * ny * sizeof(float)));
    gpuErrchk(hipMemcpy(d_data, data, nx * ny * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **)&d_sums, ny * sizeof(float)));
    gpuErrchk(hipMemset(d_sums, 0, ny * sizeof(float)));

    gpuErrchk(hipMalloc((void **)&d_result, ny * ny * sizeof(float)));

    dim3 dimGrid((nx + 2 * BLOCK_DIM - 1) / (2 * BLOCK_DIM), ny);

    get_sum<<<dimGrid, BLOCK_DIM>>>(d_data, d_sums, nx);
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());
    
    
    dim3 grid_dim((ny+31)/32, (ny+31)/32);
    dim3 block_dim(32,32);
    corr<<<grid_dim, block_dim>>>(ny, nx, d_data, d_result, d_sums);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(result, d_result, ny * ny * sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(d_data));
    gpuErrchk(hipFree(d_result));
    gpuErrchk(hipFree(d_sums));
}