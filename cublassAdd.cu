#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cmath>  // For fabs()

#define EPSILON 1e-5  // Small tolerance for floating-point comparisons

// Function to check cuBLAS API call results
#define CHECK_CUBLAS(call) { \
    hipblasStatus_t err = call; \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        printf("cuBLAS error at %s:%d, error code: %d\n", __FILE__, __LINE__, err); \
        exit(1); \
    } \
}

// Function to verify results
void verify_result(float* a, float* b, float* c, float factor, int n) {
    for (int i = 0; i < n; i++) {
        float expected = factor * a[i] + b[i];  // Correct formula
        if (fabs(c[i] - expected) > EPSILON) {
            printf("Mismatch at index %d: got %f, expected %f (factor = %f, a[i] = %f, b[i] = %f)\n",
                   i, c[i], expected, factor, a[i], b[i]);
            assert(fabs(c[i] - expected) < EPSILON);  // Assertion with tolerance
        }
    }
    printf("Verification passed!\n");
}

int main() {
    int n = 1 << 10;  // 1024 elements
    size_t bytes = n * sizeof(float);
    float factor = 2.0f;  // Scaling factor for SAXPY

    // Allocate host memory
    float *a = (float*) malloc(bytes);
    float *b = (float*) malloc(bytes);
    float *c = (float*) malloc(bytes);

    // Initialize host arrays with random values
    for (int i = 0; i < n; i++) {
        a[i] = (float)(rand() % 100);  // Corrected type casting
        b[i] = (float)(rand() % 100);
    }

    // Debugging: Print first few values before SAXPY
    printf("Factor: %f\n", factor);
    for (int i = 0; i < 5; i++) {
        printf("Before SAXPY: a[%d] = %f, b[%d] = %f\n", i, a[i], i, b[i]);
    }

    // Allocate device memory
    float *d_a, *d_b;
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);

    // Copy data from host to device
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Perform SAXPY operation: d_b = factor * d_a + d_b
    CHECK_CUBLAS(hipblasSaxpy(handle, n, &factor, d_a, 1, d_b, 1));

    // Debugging: Check device values after SAXPY
    float* temp_b = (float*)malloc(bytes);
    hipMemcpy(temp_b, d_b, bytes, hipMemcpyDeviceToHost);
    for (int i = 0; i < 5; i++) {
        printf("After SAXPY (Device): b[%d] = %f\n", i, temp_b[i]);
    }
    free(temp_b);

    // Copy result back to host
    hipMemcpy(c, d_b, bytes, hipMemcpyDeviceToHost);

    // Verify the result
    verify_result(a, b, c, factor, n);

    // Clean up
    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    free(a);
    free(b);
    free(c);

    printf("Program completed successfully!\n");
    return 0;
}
