#include <complex.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <hipfft/hipfft.h>

void runFFT(hipfftHandle &plan, hipStream_t &stream, std::vector<std::complex<float>>&signal, int fft_size, int batch){

    hipfftComplex *gpu_data = nullptr;

    // malloc
    hipMalloc(reinterpret_cast<void **>(&gpu_data), sizeof(std::complex<float>) * signal.size());
    hipMemcpyAsync(gpu_data, signal.data(), sizeof(std::complex<float>) * signal.size(), hipMemcpyHostToDevice, stream);

    // forward fft: transforms time doimain input to frequency domain
    hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_FORWARD);


    hipMemcpyAsync(signal.data(), gpu_data, sizeof(std::complex<float>) * signal.size(), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    hipFree(gpu_data);
}



int main(){
    
    hipfftHandle plan;
    hipStream_t stream = NULL;

    int fft_size = 8;
    int batch = 2;
    int num_elements = batch * fft_size;

    using scalar_type = float;
    using data_type = std::complex<scalar_type>;

    // initializing complex values
    std::vector<data_type> signal(num_elements, 0);

    for(int i=0; i<num_elements; i++){
        signal[i] = data_type(i, -i);
    }

    printf("Input array:\n");
    for(auto &i : signal){
        printf("%f + %fj\n", i.real(), i.imag());
    }


    // we need a cuFFT plan
    hipfftCreate(&plan);
    hipfftPlan1d(&plan, fft_size, HIPFFT_C2C, batch);

    // set up cuda stream
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipfftSetStream(plan, stream);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start, stream);

    runFFT(plan, stream, signal, fft_size, batch);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms\n";

    printf("Output: \n");
    for(auto &i : signal){
        printf("%f + %fj\n", i.real(), i.imag());
    }


    // destroy handle
    hipfftDestroy(plan);
    hipStreamDestroy(stream);
    hipDeviceReset();

    return 0;

}



