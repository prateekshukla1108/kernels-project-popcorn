#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "utils.cuh"

/*
Vectorized Sgemv kernel

- Each block is assigned to a row of the matrix A
- Each block calculates one output element of y
- The columns are accessed in coalesced manner by threads
- Vectorized loads are done for efficient memory bandwidth
- Performs warp level + block level sum reduction
*/
__global__ void vectorized_sgemv_kernel(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N) {
    extern __shared__ float smem[];

    int bid = blockIdx.x;
    if (bid >= M) return;

    int tid = threadIdx.x;
    int n_float4s = N / 4;

    // cast the matrix and vector as float4
    // float4 holds multiple values (x, y, z, w)
    float4* mat_row = reinterpret_cast<float4*>(matd + bid * N);
    float4* vec = reinterpret_cast<float4*>(vecd);

    // each thread calculates its own partial output
    float partial_sum = 0.f;

// manual loop unrolling with a factor of 4
#pragma unroll 4
    for (int col = tid; col < n_float4s; col += blockDim.x) {
        float4 matval = mat_row[col];
        float4 vecval = vec[col];

        partial_sum += (matval.x * vecval.x +
                        matval.y * vecval.y +
                        matval.z * vecval.z +
                        matval.w * vecval.w);
    }

    // block level sum reduction
    // only first thread reads the first location in shared memory
    // only first thread writes the output to global memory
    blockReduceSum(partial_sum, smem, tid, blockDim.x);
    if (tid == 0) {
        float sum = smem[0];
        resd[bid] = sum;
    }
}

/*
Runs the vectorized sgemv kernel.
*/
float run_kernel_vectorized_sgmev(float* __restrict__ matd, float* __restrict__ vecd, float* __restrict__ resd, int M, int N, float THEORETICAL_MAX_GFLOPS, float THEORETICAL_MAX_MEMORY_BANDWIDTH) {
    int NUM_THREADS = 64;
    int warp_size = 32;

    dim3 block_size(NUM_THREADS);
    dim3 grid_size(M);
    size_t shared_mem_size = CEIL_DIV(block_size.x, warp_size) * sizeof(float);

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    float ms = 0.f;

    CUDA_CHECK(hipEventRecord(start));
    vectorized_sgemv_kernel<<<grid_size, block_size, shared_mem_size>>>(matd, vecd, resd, M, N);
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    printf("------- Vectorized sgmev kernel ---------\n");
    print_kernel_essentials(M, N, ms, THEORETICAL_MAX_GFLOPS, THEORETICAL_MAX_MEMORY_BANDWIDTH);
    printf("---------------------------\n");

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return ms;
}
