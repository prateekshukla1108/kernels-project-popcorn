#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel to normalize an array
__global__ void normalizeKernel(float *d_out, float sum, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        d_out[idx] /= sum;  // Normalize each element
    }
}

int main() {
    const int N = 5;  
    float h_out[N] = {2.0, 4.0, 6.0, 8.0, 10.0}; // Example input
    float h_sum = 30.0;  // Example sum

    float *d_out;
    hipMalloc((void**)&d_out, N * sizeof(float));
    hipMemcpy(d_out, h_out, N * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    normalizeKernel<<<numBlocks, blockSize>>>(d_out, h_sum, N);
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Normalized Output:\n");
    for (int i = 0; i < N; i++) {
        printf("%f ", h_out[i]);
    }
    printf("\n");

    hipFree(d_out);
    return 0;
}

