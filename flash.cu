#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>  
#include <limits> 
#include "helper.cuh"
#include "kernels.cuh"

#define BLOCK_SIZE 1024
#define THREADS_PER_BLOCK 1024
#define NEGATIVE_INFINITY -1e38f

void flashAttention2BackwardPass(const float* Q, const float* K, const float* V, const float* O, const float* dO, float* dQ, float* dK, float* dV, int N, int d, int Bc, int Br, float* Lhost) {
    float scale = 1.0f / sqrtf((float)d);
    // Initialize D
    float* D_device;
    hipMalloc((void**)&D_device, N * sizeof(float));
    computeDKernel<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dO, O, D_device, N, d);
    hipDeviceSynchronize();

    float* D_host = (float*)malloc(N * sizeof(float));
    hipMemcpy(D_host, D_device, N * sizeof(float), hipMemcpyDeviceToHost);

    // Initialize dQ, dK, dV on device
    hipMemset(dQ, 0, N * d * sizeof(float));
    hipMemset(dK, 0, N * d * sizeof(float));
    hipMemset(dV, 0, N * d * sizeof(float));

    for (int j = 0; j < (N + Bc - 1) / Bc; ++j) {
        // Load Kj, Vj from HBM to on-chip SRAM (Simulated by host memory for now)
        float* Kj_host = (float*)malloc(Bc * d * sizeof(float));
        float* Vj_host = (float*)malloc(Bc * d * sizeof(float));
        hipMemcpy(Kj_host, K + j * Bc * d, Bc * d * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(Vj_host, V + j * Bc * d, Bc * d * sizeof(float), hipMemcpyDeviceToHost);

        float* Kj_device;
        float* Vj_device;
        hipMalloc((void**)&Kj_device, Bc * d * sizeof(float));
        hipMalloc((void**)&Vj_device, Bc * d * sizeof(float));
        hipMemcpy(Kj_device, Kj_host, Bc * d * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(Vj_device, Vj_host, Bc * d * sizeof(float), hipMemcpyHostToDevice);

        // Initialize dKj, dVj on SRAM (Simulated by device memory for now)
        float* dKj_temp;
        float* dVj_temp;
        hipMalloc((void**)&dKj_temp, Bc * d * sizeof(float));
        hipMalloc((void**)&dVj_temp, Bc * d * sizeof(float));
        hipMemset(dKj_temp, 0, Bc * d * sizeof(float));
        hipMemset(dVj_temp, 0, Bc * d * sizeof(float));

        for (int i = 0; i < (N + Br - 1) / Br; ++i) {
            // Load Qi, dOi, dQi, Li, Di from HBM to on-chip SRAM (Simulated by device memory for now)
            const float* Qi = Q + i * Br * d;
            const float* dOi = dO + i * Br * d;
            float* dQi_temp;
            hipMalloc((void**)&dQi_temp, Br * d * sizeof(float));
            hipMemset(dQi_temp, 0, Br * d * sizeof(float));

            const float* Li = Lhost + i * Br; // Assuming L is divided into blocks of size Br
            const float* Di = D_host + i * Br; // D is now divided into blocks of size Br

            // Allocate intermediate buffers on device for each loop iteration
            float* Si_device;
            float* Pi_device;
            float* dPi_device;
            float* dSi_device;
            hipMalloc((void**)&Si_device, Br * Bc * sizeof(float));
            hipMalloc((void**)&Pi_device, Br * Bc * sizeof(float));
            hipMalloc((void**)&dPi_device, Br * Bc * sizeof(float));
            hipMalloc((void**)&dSi_device, Br * Bc * sizeof(float));

            // Compute S_i
            computeSiKernel<<<(Br + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(Qi, Kj_device, Si_device, Br, Bc, d, scale);
            hipDeviceSynchronize();

            // Find row-wise max of S_i
            float* maxSi_device;
            hipMalloc((void**)&maxSi_device, Br * sizeof(float));
            findRowMaxSiKernel<<<(Br + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(Si_device, maxSi_device, Br, Bc);
            hipDeviceSynchronize();

            // Compute P_i
            computePiKernel<<<(Br + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(Si_device, Li, Pi_device, Br, Bc, maxSi_device);
            hipDeviceSynchronize();

            // Compute dVj += (P_i^T) * dOi
            computeDViKernel<<<(d + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(Pi_device, dOi, dVj_temp, Br, Bc, d);
            hipDeviceSynchronize();

            // Compute dPi = dOi * V_j^T
            computeDPiKernel<<<(Br + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dOi, Vj_device, dPi_device, Br, Bc, d);
            hipDeviceSynchronize();

            // Compute dS_i = P_i * (dP_i - D_i)
            computeDSiKernel<<<(Br + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(Pi_device, dPi_device, Di, dSi_device, Br, Bc);
            hipDeviceSynchronize();

            // Compute dQ_i += dS_i * K_j
            computeDQiKernel<<<(Br + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dSi_device, Kj_device, dQi_temp, Br, d, Bc);
            hipDeviceSynchronize();

            // Compute dKj += dS_i^T * Q_i
            computeDKjKernel<<<(Bc + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dSi_device, Qi, dKj_temp, Bc, d, Br);
            hipDeviceSynchronize();

            // Accumulate into dQ
            accumulateDQKernel<<<(Br * d + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dQ, dQi_temp, Br, d, i * Br * d);
            hipDeviceSynchronize();

            // Free intermediate buffers
            hipFree(Si_device);
            hipFree(Pi_device);
            hipFree(dPi_device);
            hipFree(dSi_device);
            hipFree(maxSi_device);
            hipFree(dQi_temp);
        }

        // Accumulate into dK and dV
        accumulateDKVjKernel<<<(Bc * d + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(dK, dV, dKj_temp, dVj_temp, Bc, d, j * Bc * d);
        hipDeviceSynchronize();

        // Free device memory for Kj, Vj, dKj, dVj
        hipFree(Kj_device);
        hipFree(Vj_device);
        hipFree(dKj_temp);
        hipFree(dVj_temp);
        free(Kj_host);
        free(Vj_host);
    }

    hipFree(D_device);
    free(D_host);
}