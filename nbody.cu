#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <cmath>

#define BLOCK_SIZE 128
#define G 1.0f
#define SOFTENING 1e-9f

struct Body
{
    float3 pos;
    float3 vel;
    float mass;
};

__device__ inline float3 addFloat3(const float3 &a, const float3 &b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}


__device__ inline float3 subFloat3(const float3 &a, const float3 &b)
{

    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);

}

__device__ inline float3 scaleFloat3(const float3 &a, float s)

{
    
    return make_float3(a.x * s, a.y * s, a.z * s);


}

__global__ void nbodyKernel(const Body *bodies, Body *new_bodies, int n, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Body myBody = bodies[i];
    float3 acc = make_float3(0.0f, 0.0f, 0.0f);
    extern __shared__ Body sharedBodies[];
    int numTiles = (n + blockDim.x - 1) / blockDim.x;

    for (int tile = 0; tile < numTiles; tile++)
    {
        int idx = tile * blockDim.x + threadIdx.x;
        if (idx < n)
        {
            sharedBodies[threadIdx.x] = bodies[idx];
        }
        else
        {
            sharedBodies[threadIdx.x].pos = make_float3(0.0f, 0.0f, 0.0f);
            sharedBodies[threadIdx.x].vel = make_float3(0.0f, 0.0f, 0.0f);
            sharedBodies[threadIdx.x].mass = 0.0f;
        }
        __syncthreads();

#pragma unroll
        for (int j = 0; j < blockDim.x; j++)
        {
            int globalIndex = tile * blockDim.x + j;
            if (globalIndex < n && globalIndex != i)
            {
                float3 r = subFloat3(sharedBodies[j].pos, myBody.pos);
                float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + SOFTENING;
                float invDist = rsqrtf(distSqr);
                float invDist3 = invDist * invDist * invDist;
                float factor = sharedBodies[j].mass * invDist3;
                acc.x += r.x * factor;
                acc.y += r.y * factor;
                acc.z += r.z * factor;
            }
        }
        __syncthreads();
    }

    acc = scaleFloat3(acc, G);
    myBody.vel = addFloat3(myBody.vel, scaleFloat3(acc, dt));
    myBody.pos = addFloat3(myBody.pos, scaleFloat3(myBody.vel, dt));
    new_bodies[i] = myBody;
}

void checkCuda(hipError_t err, const char *msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main()
{
    const int nBodies = 1 << 14;
    const size_t bytes = nBodies * sizeof(Body);
    const float dt = 0.01f;

    Body *h_bodies = (Body *)malloc(bytes);
    Body *h_new_bodies = (Body *)malloc(bytes);

    for (int i = 0; i < nBodies; i++)
    {
        h_bodies[i].pos = make_float3(
            (float)rand() / RAND_MAX,
            (float)rand() / RAND_MAX,
            (float)rand() / RAND_MAX);
        h_bodies[i].vel = make_float3(
            ((float)rand() / RAND_MAX) * 0.1f,
            ((float)rand() / RAND_MAX) * 0.1f,
            ((float)rand() / RAND_MAX) * 0.1f);
        h_bodies[i].mass = ((float)rand() / RAND_MAX) + 0.1f;
    }

    Body *d_bodies, *d_new_bodies;
    checkCuda(hipMalloc(&d_bodies, bytes), "Allocating d_bodies");
    checkCuda(hipMalloc(&d_new_bodies, bytes), "Allocating d_new_bodies");
    checkCuda(hipMemcpy(d_bodies, h_bodies, bytes, hipMemcpyHostToDevice), "Copying bodies to device");

    int blockSize = BLOCK_SIZE;
    int gridSize = (nBodies + blockSize - 1) / blockSize;
    size_t sharedMemSize = blockSize * sizeof(Body);

    nbodyKernel<<<gridSize, blockSize, sharedMemSize>>>(d_bodies, d_new_bodies, nBodies, dt);
    checkCuda(hipGetLastError(), "Kernel launch");
    checkCuda(hipDeviceSynchronize(), "Kernel execution");
    checkCuda(hipMemcpy(h_new_bodies, d_new_bodies, bytes, hipMemcpyDeviceToHost), "Copying new bodies to host");

    for (int i = 0; i < 5; i++)
    {
        printf("Body %d: pos = (%f, %f, %f)\n", i,
               h_new_bodies[i].pos.x,
               h_new_bodies[i].pos.y,
               h_new_bodies[i].pos.z);
    }

    free(h_bodies);
    free(h_new_bodies);
    hipFree(d_bodies);
    hipFree(d_new_bodies);

    return 0;
}
