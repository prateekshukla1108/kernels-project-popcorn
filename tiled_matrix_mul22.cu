#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cassert>
#include "../helpers/cuda_helpers.h"

#define TILE_WIDTH 8

void initializeMatrices(float* A, float* B, int M, int K, int N);



// P = M * N
__global__ void tiledMatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Saved as automatic variables thus in registers. 
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    // As we declare this variable as automatic it will be private for each thread!
    float Pvalue = 0;
    for (int phase = 0; phase < Width / TILE_WIDTH; ++phase){
        // Collaborative loading of d_M and d_N tiles into shared memory
        Mds[ty][tx] = d_M[Row * Width + phase * TILE_WIDTH + tx];
        Nds[ty][tx] = d_N[(phase * TILE_WIDTH + threadIdx.y) * Width + Col]; 
        __syncthreads();

        for (int k =0; k < TILE_WIDTH; ++k){
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();
    }
    d_P[Row * Width + Col] = Pvalue;
}

__global__ void simpleMatrixMulKernel(float* M, float* N, float* P, int width){
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    if ((Row < width) && (Col < width)){
        float Pvalue = 0;
        for (int i = 0; i < width; i++){
            // Since here we access element of M and one element of N
            // and we apply one addition and one multiplication.
            // compute to global-memory-access ratio is 1. 
            // But we need to increase this ratio!
            Pvalue += M[Row * width + i] * N[i * width + Col];
        }
        P[Row * width + Col] = Pvalue;
    }
}



void initializeMatrices(float* A, float* B, int M, int K, int N) {
    for(int i = 0; i < M * K; i++) A[i] = rand() / (float)RAND_MAX;
    for(int i = 0; i < K * N; i++) B[i] = rand() / (float)RAND_MAX;
}

void testTiledMatrixMul() {
    int size = 16;
    int matrixSize = size * size * sizeof(float);
    float *h_M = (float*)malloc(matrixSize);
    float *h_N = (float*)malloc(matrixSize);

    initializeMatrices(h_M, h_N, size, size, size);

    // printArray(h_M, size, size, "Matrix M");
    // printArray(h_N, size, size, "Matrix N");

    float h_P_1[size * size];
    float h_P_2[size * size];

    float *d_M, *d_N, *d_P_1, *d_P_2;

    hipMalloc((void**)&d_M, matrixSize);
    hipMalloc((void**)&d_N, matrixSize);
    hipMalloc((void**)&d_P_1, matrixSize);
    hipMalloc((void**)&d_P_2, matrixSize);


    hipMemcpy(d_M, h_M, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, matrixSize, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((size + TILE_WIDTH - 1) / TILE_WIDTH, (size + TILE_WIDTH - 1) / TILE_WIDTH);

    hipEvent_t start, stop;
    float elapsedTime;

    // Timing for tiledMatrixMulKernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    tiledMatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P_1, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Tiled Matrix Multiplication Time: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Timing for simpleMatrixMulKernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    simpleMatrixMulKernel<<<dimGrid, dimBlock>>>(d_M, d_N, d_P_2, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Simple Matrix Multiplication Time: " << elapsedTime << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_P_1, d_P_1, matrixSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_P_2, d_P_2, matrixSize, hipMemcpyDeviceToHost);

    // uncomment to print arrays
    // printArray(h_M, size, size, "Matrix M");
    // printArray(h_N, size, size, "Matrix N");
    printArray(h_P_1, size, size, "Matrix P_1 TILED");
    printArray(h_P_2, size, size, "Matrix P_2 NAIVE");


    for (int i = 0; i < size * size; i++) {
        assert(fabs(h_P_1[i] - h_P_2[i]) < 1e-5);
    }

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P_1);
    hipFree(d_P_2);
}

int main() {
    testTiledMatrixMul();
    return 0;
}
