#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16 

__global__ void matrixMul(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N *N * sizeof(int);
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);



    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1;
        h_B[i] = 1;
    }

    // Allocate memory on the device
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    matrixMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


    printf("C[0][0] = %d\n", h_C[0]);


    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
