// tiled matrix multiplication

#include <iostream>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 16
#define N 1024

__global__  void tiledMatMul(float *d_M, float *d_N, float *d_P, int width){
    
    // shared variables
    __shared__ float M_shared[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_shared[TILE_WIDTH][TILE_WIDTH];

    // automatic variables saved into registers
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // index
    int row = by*TILE_WIDTH + ty;
    int col = bx*TILE_WIDTH + tx;

    // automatic variable
    float P = 0;

    // iterate through phases to compute P
    int max_phases = width / TILE_WIDTH;
    for (int phase = 0; phase < max_phases; ++phase){

        // load each tile element into shared memory
        M_shared[ty][tx] = d_M[row*width + phase*TILE_WIDTH + tx];
        N_shared[ty][tx] = d_N[(phase*TILE_WIDTH + ty) * width + col];

        __syncthreads(); // wait for all threads to finish loading tiles

        // each thread computes an element of the output matrix
        for (int k = 0; k < TILE_WIDTH; ++k){
            P += M_shared[ty][k] * N_shared[k][tx]; 
        }
        __syncthreads();
    }

    d_P[row*width + col] = P;
}


int main(){

    size_t matrix_size = N*N*sizeof(float);

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    h_A = (float*)malloc(matrix_size);
    h_B = (float*)malloc(matrix_size);
    h_C = (float*)malloc(matrix_size);

    hipMalloc((void**)&d_A, matrix_size);
    hipMalloc((void**)&d_B, matrix_size);
    hipMalloc((void**)&d_C, matrix_size);

    for(int i=0; i<N*N; i++){
        h_A[i] = (i % N) + 1;
        h_B[i] = (i / N) + 1;
    }

    hipMemcpy(d_A, h_A, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrix_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 blocksPerGrid(N/TILE_WIDTH, N/TILE_WIDTH);

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    tiledMatMul<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Record stop event
    hipEventRecord(stop);
    hipEventSynchronize(stop);  // Ensure kernel execution is finished

    // Compute elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms\n";


    hipMemcpy(h_C, d_C, matrix_size, hipMemcpyDeviceToHost);

    printf("Successful");
    printf("Sample result C[0][0] = %f\n", h_C[0]);

    // free memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}