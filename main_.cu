#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <cmath>

__global__ void vectorAddRuntime(const float *A, const float *B, float *C, unsigned int N) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    const unsigned int N = 1024 * 1024;
    const size_t memSize = N * sizeof(float);

    std::vector<float> h_A(N), h_B(N), h_C_ptx(N), h_C_runtime(N);
    for (unsigned int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(2 * i);
    }

    hipError_t cudaErr = hipFree(0);
    if (cudaErr != hipSuccess) {
        std::cerr << "cudaFree(0) failed: " << hipGetErrorString(cudaErr) << std::endl;
        return -1;
    }

    hipError_t res;
    hipDevice_t device;
    hipCtx_t context;
    res = hipInit(0);
    if (res != hipSuccess) {
        std::cerr << "hipInit failed" << std::endl;
        return -1;
    }
    res = hipDeviceGet(&device, 0);
    if (res != hipSuccess) {
        std::cerr << "hipDeviceGet failed" << std::endl;
        return -1;
    }
    res = hipDevicePrimaryCtxRetain(&context, device);
    if (res != hipSuccess) {
        std::cerr << "hipDevicePrimaryCtxRetain failed" << std::endl;
        return -1;
    }

    hipModule_t module;
    res = hipModuleLoad(&module, "day23/kernel.ptx");
    if (res != hipSuccess) {
        std::cerr << "Failed to load PTX module" << std::endl;
        return -1;
    }

    hipFunction_t ptxKernel;
    res = hipModuleGetFunction(&ptxKernel, module, "vectorAdd");
    if (res != hipSuccess) {
        std::cerr << "Failed to get kernel function from PTX module" << std::endl;
        return -1;
    }

    hipDeviceptr_t d_A_driver, d_B_driver, d_C_driver;
    res = hipMalloc(&d_A_driver, memSize);
    res = hipMalloc(&d_B_driver, memSize);
    res = hipMalloc(&d_C_driver, memSize);

    res = hipMemcpyHtoD(d_A_driver, h_A.data(), memSize);
    res = hipMemcpyHtoD(d_B_driver, h_B.data(), memSize);

    unsigned int threadsPerBlock = 256;
    unsigned int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    void *ptxKernelParams[] = { &d_A_driver, &d_B_driver, &d_C_driver, (void *)&N };

    hipEvent_t start_ptx, stop_ptx;
    hipEventCreateWithFlags(&start_ptx, hipEventDefault);
    hipEventCreateWithFlags(&stop_ptx, hipEventDefault);
    hipEventRecord(start_ptx, 0);

    res = hipModuleLaunchKernel(ptxKernel,
                         blocks, 1, 1,
                         threadsPerBlock, 1, 1,
                         0,
                         0,
                         ptxKernelParams,
                         0);
    if (res != hipSuccess) {
        std::cerr << "Failed to launch PTX kernel" << std::endl;
        return -1;
    }

    hipEventRecord(stop_ptx, 0);
    hipEventSynchronize(stop_ptx);
    float time_ptx;
    hipEventElapsedTime(&time_ptx, start_ptx, stop_ptx);
    std::cout << "PTX kernel execution time: " << time_ptx << " ms" << std::endl;

    res = hipMemcpyDtoH(h_C_ptx.data(), d_C_driver, memSize);

    float *d_A_runtime, *d_B_runtime, *d_C_runtime;
    hipMalloc(&d_A_runtime, memSize);
    hipMalloc(&d_B_runtime, memSize);
    hipMalloc(&d_C_runtime, memSize);

    hipMemcpy(d_A_runtime, h_A.data(), memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B_runtime, h_B.data(), memSize, hipMemcpyHostToDevice);

    hipEvent_t start_rt, stop_rt;
    hipEventCreate(&start_rt);
    hipEventCreate(&stop_rt);
    hipEventRecord(start_rt, 0);

    vectorAddRuntime<<<blocks, threadsPerBlock>>>(d_A_runtime, d_B_runtime, d_C_runtime, N);

    hipEventRecord(stop_rt, 0);
    hipEventSynchronize(stop_rt);
    float time_rt;
    hipEventElapsedTime(&time_rt, start_rt, stop_rt);
    std::cout << "Runtime kernel execution time: " << time_rt << " ms" << std::endl;

    hipMemcpy(h_C_runtime.data(), d_C_runtime, memSize, hipMemcpyDeviceToHost);

    bool success = true;
    for (unsigned int i = 0; i < N; ++i) {
        float expected = h_A[i] + h_B[i];
        if (fabs(h_C_ptx[i] - expected) > 1e-5 || fabs(h_C_runtime[i] - expected) > 1e-5) {
            std::cerr << "Mismatch at index " << i << std::endl;
            success = false;
            break;
        }
    }
    if (success) {
        std::cout << "Both kernels produced correct results." << std::endl;
    }

    hipFree(d_A_driver);
    hipFree(d_B_driver);
    hipFree(d_C_driver);
    hipFree(d_A_runtime);
    hipFree(d_B_runtime);
    hipFree(d_C_runtime);
    hipEventDestroy(start_ptx);
    hipEventDestroy(stop_ptx);
    hipEventDestroy(start_rt);
    hipEventDestroy(stop_rt);
    hipModuleUnload(module);
    hipDevicePrimaryCtxRelease(device);

    return 0;
}
