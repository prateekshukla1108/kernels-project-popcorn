#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE (1 << 20) 


__global__ void geluKernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        float cdf = 0.5f * (1.0f + tanhf(0.7978845608f * (x + 0.044715f * x * x * x))); 
        output[idx] = x * cdf;
    }
}

void geluCPU(const float* input, float* output, int size) {
    for (int i = 0; i < size; i++) {
        float x = input[i];
        float cdf = 0.5f * (1.0f + tanhf(0.7978845608f * (x + 0.044715f * x * x * x))); 
        output[i] = x * cdf;
    }
}
int main() {
    int size = ARRAY_SIZE;
    size_t arraySize = size * sizeof(float);
    
    float *h_input = (float*)malloc(arraySize);
    float *h_output_cpu = (float*)malloc(arraySize);
    float *h_output_gpu = (float*)malloc(arraySize);
    
    for (int i = 0; i < size; i++) {
        h_input[i] = 2.0f * (static_cast<float>(rand()) / RAND_MAX - 0.5f); 
    }
    
    float *d_input, *d_output;
    hipMalloc(&d_input, arraySize);
    hipMalloc(&d_output, arraySize);
    
    hipMemcpy(d_input, h_input, arraySize, hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    geluKernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpuTime = 0.0f;
    hipEventElapsedTime(&gpuTime, start, stop);
    
    hipMemcpy(h_output_gpu, d_output, arraySize, hipMemcpyDeviceToHost);
    
    clock_t cpuStart = clock();
    geluCPU(h_input, h_output_cpu, size);
    clock_t cpuEnd = clock();
    float cpuTime = 1000.0f * (cpuEnd - cpuStart) / CLOCKS_PER_SEC; 

    
    bool correct = true;
    for (int i = 0; i < size; i++) {
        if (fabs(h_output_cpu[i] - h_output_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }
    if (correct) {
        printf("Results match! GELU implementation is correct.\n");
    } else {
        printf("Results do not match! Check the implementation.\n");
    }
    
    printf("CPU Time: %.3f ms\n", cpuTime);
    printf("GPU Time: %.3f ms\n", gpuTime);
    printf("Speedup: %.2fx\n", cpuTime / gpuTime);
    
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output_cpu);
    free(h_output_gpu);
    return 0;
}