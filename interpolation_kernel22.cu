#include "hip/hip_runtime.h"
torch::Tensor trilinear_fwd_cuda(torch::Tensor feats, torch::Tensor points)
{
    CHECK_INPUT(feats);
    CHECK_INPUT(points);

    const int N = feats.size(0);
    const int F = feats.size(2);

    torch::Tensor feat_interp = torch::zeros({N, F}, feats.options());

    const dim3 threads(16, 16);
    const dim3 blocks((N + threads.x - 1) / threads.x, (F + threads.y - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fwd_cuda", ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits>(),
            feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits>()
        );
    }));

    return feat_interp;
}

template <typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits> points,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits> feat_interp)
{
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n < feats.size(0) && f < feats.size(2)) {
        const scalar_t u = (points[n][0] + 1) / 2;
        const scalar_t v = (points[n][1] + 1) / 2;
        const scalar_t w = (points[n][2] + 1) / 2;

        const scalar_t a = (1 - v) * (1 - w);
        const scalar_t b = v * (1 - w);
        const scalar_t c = v * w;
        const scalar_t d = 1 - a - b - c;

        feat_interp[n][f] = (1 - u) * (a * feats[n][0][f] +
                                      b * feats[n][1][f] +
                                      c * feats[n][2][f] +
                                      d * feats[n][3][f]) +
                              u * (a * feats[n][4][f] +
                                   b * feats[n][5][f] +
                                   c * feats[n][6][f] +
                                   d * feats[n][7][f]);
    }
} 