#include<iostream>
#include<hip/hip_runtime.h>

// matrix addition kernel
__global__  void add(float* A, float* B, float* C, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        // row-major order 
        int index = row * width + col;

        C[index] = A[index] + B[index];
    }
}

int main() {
    int width = 1024;
    int height = 1024;
    int size = width * height * sizeof(float);

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    for (int i = 0; i < width * height; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    float time = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    add<<<gridSize, blockSize>>>(d_A, d_B, d_C, width, height);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Kernel execution time: " << time << " ms" << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
// Kernel execution time: 0.356384 ms